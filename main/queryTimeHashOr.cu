#include <iostream>
#include <fstream>
#include <cstdlib>
#include <unistd.h>

#include <sparsehash/dense_hash_map>
#include <sys/time.h>

#include "header/manager.hxx"


using google::dense_hash_map;



//TODO implementare la projection su gpu
//TODO PASSARE I TIMESTAMP DA US A MS (OVERFLOW DI DAY E HOUR)


template<typename type_t, typename accuracy>
std::vector<accuracy> stats(std::vector<type_t> input) {
	int size = input.size();
	float mean = 0;
	float variance = 0;
	for (int i = 0; i < size; i++) {
		mean += (accuracy) input[i];
                variance += (accuracy)  (input[i] * input[i]);
        }
        mean = mean / ((accuracy) size);
        variance = variance / ((accuracy) size);
        variance = variance - (mean * mean);
        std::vector<accuracy> statistic;
	statistic.push_back(mean);
	statistic.push_back(variance);
	return statistic;
}



int main(int argc, char** argv) {
 
	using namespace std;

	std::vector<float> timeCuVector;                	
	std::vector<long int> timeExVector;	

	//READ STORE FROM FILE
	ifstream rdfStoreFile (argv[1]);
	string strInput;

	int fileLength = 0;	 
	while (std::getline(rdfStoreFile, strInput)) {
		++fileLength;
	}
	
	std::cout << "STORE SIZE IS " << fileLength << std::endl;
		
	rdfStoreFile.clear();
	rdfStoreFile.seekg(0, ios::beg);

	size_t rdfSize = fileLength  * sizeof(std::string);
	std::string* h_rdfStore = (std::string*) malloc(rdfSize);

	for (int i = 0; i <fileLength; i++) {
		getline(rdfStoreFile,strInput);
	    h_rdfStore[i]  = strInput;
	}
    rdfStoreFile.close();
	//END RDF READ

	struct timeval beginT, endT;
	
	hipDeviceReset();

    size_t BUFFER_SIZE = 400000;
  
    int N_CYCLE = 1;

	for (int i = 0; i < N_CYCLE; i++) {

		gettimeofday(&beginT, NULL);

		QueryManager manager(h_rdfStore, fileLength, BUFFER_SIZE);
					
		try {
			//TODO controllare errore se manca sapazio finale? 
		//	manager.parseQuery("FROM STREAM <streamUri> RANGE TRIPLES 7000 SELECT ?s WHERE { ?p ?s <http://example.org/int/9> . <http://example.org/int/90> ?w ?p } ");
			manager.parseQuery(argv[2]);
		}
		catch (std::string exc) {
			std::cout << "Exception raised: " << exc << std::endl;
			exit(1);
		}
		
		manager.start();
		hipDeviceSynchronize();
		gettimeofday(&endT, NULL);

		float exTime = (endT.tv_sec - beginT.tv_sec ) * 1000 + ((float) endT.tv_usec - (float) beginT.tv_usec) / 1000 ;
					
		timeCuVector.push_back(exTime);

		cout << "Time: " << exTime << endl;
	}

	std::vector<float> statistics = stats<float, float>(timeCuVector);	
    cout << "mean cuda time " << statistics[0] << endl;
    cout << "variance cuda time " << statistics[1] << endl;
	cout << "FINAL VALUE IS " << VALUE << std::endl;
				
    return 0;
}

