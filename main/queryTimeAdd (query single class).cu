#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <unistd.h>
#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_join.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <sys/time.h>

using namespace mgpu;

int TEST_VALUE[2]  {0, 0};

//struct to contains a single triple with int type.
struct tripleContainer {
        int subject;
        int predicate;
        int object;
};

//Struct for circular buffer
template<typename type_t>
struct circularBuffer {
	type_t* pointer;
	int begin;
	int end;
	int size;
	
	circularBuffer() : pointer(0), begin(0), end(0), size(0) {}
};

//Struct for containing the pointer to an rdf store (divided into subject predicate and object)
template<typename rdf_t, typename arr_t>
struct triplePointer {
	rdf_t rdfStore;
	arr_t subject;
	arr_t predicate;
	arr_t object;
};

/*
* Specific implementation of triplePointer for ciruclar buffer.
* Offers methods for managing the attributes of the class.
*/
struct deviceCircularBuffer : triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> {
	void setValues(int begin, int end, int size) {
		setBegin(begin);
		setEnd(end);
		setSize(size);
	}	

	void setBegin(int begin) {
		rdfStore.begin = begin;
		subject.begin = begin;
		predicate.begin = begin;
		object.begin = begin;
	}
	
	void setEnd(int end) {
		rdfStore.end = end;
		subject.end = end;
		predicate.end = end;
		object.end = end;
	}
	
	void setSize(int size) {
		rdfStore.size = size;
		subject.size = size;
		predicate.size = size;
		object.size = size;
	}
	
	void advanceBegin(int step){
		int newBegin = (rdfStore.begin + step) % rdfStore.size;
		setBegin(newBegin);
	}
	
	void advanceEnd(int step){
		int newEnd = (rdfStore.end + step) % rdfStore.size;
		setEnd(newEnd);
	}				
};

int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}

/*
* Join enum to define order and which element to join
* NJ indicates a non-join value, so it is ignored during join and sorting
* So that it improves performance avoiding uneecessary conditional expression
*/
enum class JoinMask {NJ = -1, SBJ = 0, PRE = 1, OBJ = 2};


//Section for defining operation classes
class JoinOperation 
{	
	private:
		mem_t<tripleContainer>** innerTable;
		mem_t<tripleContainer>** outerTable;
		mem_t<tripleContainer>* innerResult = 0;
		mem_t<tripleContainer>* outerResult = 0;
		
		JoinMask innerMask[3];
		JoinMask outerMask[3];

	public:
		JoinOperation(mem_t<tripleContainer>** innerTable, mem_t<tripleContainer>** outerTable, JoinMask innerMask[3], JoinMask outerMask[3]) {
			this->innerTable = innerTable;
			this->outerTable = outerTable;
			std::copy(innerMask, innerMask + 3, this->innerMask);
			std::copy(outerMask, outerMask + 3, this->outerMask);
		};
			
		mem_t<tripleContainer>** getInnerTable() {
			return this->innerTable;
		};
		
		mem_t<tripleContainer>** getOuterTable() {
			return this->outerTable;
		};
		
		JoinMask* getInnerMask() {
			return this->innerMask;
		};
		
		JoinMask* getOuterMask() {
			return this->outerMask;
		};
		
		mem_t<tripleContainer>* getInnerResult() {
			return this->innerResult;
		};
		
		void setInnerResult(mem_t<tripleContainer>* result) {
			this->innerResult = result;
		};
		
		mem_t<tripleContainer>** getInnerResultAddress() {
			return &innerResult;
		}
		
		mem_t<tripleContainer>* getOuterResult() {
			return this->outerResult;
		};
		
		void setOuterResult(mem_t<tripleContainer>* result) {
			this->outerResult = result;
		};
		
		mem_t<tripleContainer>** getOuterResultAddress() {
			return &outerResult;
		}		
};

enum class SelectArr { S = 0, P = 1, O = 2, SP = 3, SO = 4, PO = 5, SPO = 6};

class SelectOperation 
{
	private:
		mem_t<tripleContainer>* query;
		mem_t<tripleContainer>* result = 0;
		int arr;

	public:
		SelectOperation(mem_t<tripleContainer>* query, SelectArr arr) {
			this->query = query;	
			this->arr = static_cast<int> (arr);
		};

		int getArr() {
			return this-> arr;
		}
			
		mem_t<tripleContainer>* getQuery() {
			return this->query;
		};
		                                                                            
		mem_t<tripleContainer>* getResult() {
			return this->result;
		};
		
		void setResult(mem_t<tripleContainer>* result) {
			this->result = result;
		};
		
		mem_t<tripleContainer>** getResultAddress() {
			return &result;
		}
};

std::vector<mem_t<tripleContainer>*> rdfJoin(tripleContainer* innerTable, int innerSize, tripleContainer* outerTable, int outerSize, JoinMask innerMask[3], JoinMask outerMask[3]);

std::vector<mem_t<tripleContainer>*> rdfSelect(const std::vector<tripleContainer*> d_selectQueries, 
		deviceCircularBuffer d_pointer,
		const int storeSize, 
		std::vector<int>  arrs);

class Query {
	public:
		std::vector<SelectOperation*> select;
		std::vector<JoinOperation*> join;
		deviceCircularBuffer windowPointer;
		long int lastTimestamp;

		Query(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join, deviceCircularBuffer rdfPointer) {
			this->join = join;
			this->select = select;
			this->windowPointer = rdfPointer;
		}

		virtual void advancePointer(int step) {
			windowPointer.advanceEnd(step);
		}
		
		void printResults() {
			int i = 0;
			for (auto op : select) {
				std::vector<tripleContainer> selectResults = from_mem(*(op->getResult()));
				std::cout <<"selct size " << selectResults.size() << std::endl;
				
				hipFree(op->getResult()->data());
				
				if (i <= 1) {
					TEST_VALUE[i] += selectResults.size();
				}
				
				i++;
			}
			
			for (auto op : join) {
				hipFree(op->getInnerResult()->data());
				hipFree(op->getOuterResult()->data());
			}
					
		}
		
		void setStartingTimestamp(long int timestamp) {
			this->lastTimestamp = timestamp;
		}

		
		virtual void launch() =0;
		virtual bool isReady() =0;
		
		/**
		* Function for managing query execution
		**/
		void startQuery() {
			int storeSize =  (abs(windowPointer.rdfStore.end - windowPointer.rdfStore.begin +  windowPointer.rdfStore.size) % windowPointer.rdfStore.size);			
			std::vector<tripleContainer*> d_selectQueries;
			std::vector<int> arrs;
	
			for (int i = 0; i < select.size(); i++) {
				d_selectQueries.push_back(select[i]->getQuery()->data());
				arrs.push_back(select[i]->getArr());
			}
	
			std::vector<mem_t<tripleContainer>*> selectResults = rdfSelect(d_selectQueries, windowPointer, storeSize, arrs);

			for (int i = 0; i < selectResults.size(); i++) {
				select[i]->setResult(selectResults[i]);
			}
	
	
			for (int i = 0; i < join.size(); i++) {
				mem_t<tripleContainer>* innerTable = *join[i]->getInnerTable();
				mem_t<tripleContainer>* outerTable = *join[i]->getOuterTable();
				std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin(innerTable->data(), innerTable->size(), outerTable->data(), outerTable->size(), join[i]->getInnerMask(), join[i]->getOuterMask());
				join[i]->setInnerResult(joinResult[0]);
				join[i]->setOuterResult(joinResult[1]);				
			}
	
		}

};


class CountQuery : public Query {
	private:
		int count;
		int currentCount;

	public:
		CountQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer,
			int count) : Query(select, join, rdfPointer) {

				this->count = count;
				this->currentCount = 0;
		}
		
		bool isReady() {
			this->currentCount++;
			if (currentCount == count) {
				currentCount = 0;
				return true;
			}
			return false;
		}
		

	
		void launch() {
			startQuery();
			windowPointer.advanceBegin(count);
			printResults();
		}
		
		~CountQuery() {}
};

class TimeQuery : public Query {
	private:
		circularBuffer<long int> timestampPointer;
		long int stepTime;
		long int windowTime;

	public:
		TimeQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer, circularBuffer<long int> timestampPointer,
			int windowTime, int stepTime) : Query(select, join, rdfPointer) {
				this->stepTime = stepTime;
				this->windowTime = windowTime;
				this->lastTimestamp = 0;
				this->timestampPointer = timestampPointer;
		}
		

		void advancePointer(int step)  {
			Query::advancePointer(step);
			timestampPointer.end = (timestampPointer.end + step) % timestampPointer.size;
		}

		bool isReady() {
			long int newTimestamp = timestampPointer.pointer[timestampPointer.end -1 ];
	
			if (lastTimestamp + windowTime <= newTimestamp) {
				return true;
			}

			return false;
		}

		void launch() {
			
			int newBegin = 0;
			
			for(int i = timestampPointer.begin; i  != timestampPointer.end; i = (i + 1) % timestampPointer.size) {
				if (timestampPointer.pointer[i] > lastTimestamp) {
					newBegin = i;
					break;
				}				
			}				

			windowPointer.setBegin(newBegin);
			timestampPointer.begin = newBegin;
	
			windowPointer.setEnd(windowPointer.rdfStore.end -1);

			startQuery();
			printResults();
			
			windowPointer.setEnd(windowPointer.rdfStore.end + 1);
			
			lastTimestamp += stepTime;
		}

		~TimeQuery() {}
};




class TripleGenerator {
	private:
		int spanTime;
		tripleContainer* source;
		int srcSize;
		
		std::vector<tripleContainer> rdfBuffer;
		std::vector<int> subjectBuffer;
		std::vector<int> predicateBuffer;
		std::vector<int> objectBuffer;

		std::vector<Query*> queries;
		std::vector<TimeQuery> timeQueries;
		std:.vector<CountQuery> countQueries;
		
		circularBuffer<long int> timestampPointer;
		deviceCircularBuffer devicePointer;

	public:
		TripleGenerator(tripleContainer* source, int srcSize, int spanTime,int buffSize)   {
			this->spanTime = spanTime;
			this->srcSize = srcSize;
			this->source = source;
			long int* timestamp = (long int*) malloc(buffSize * sizeof(long int));
			timestampPointer.pointer = timestamp;
			timestampPointer.size = buffSize;
		}
		
		void setDevicePointer(deviceCircularBuffer devicePointer) {
			this->devicePointer = devicePointer;
		}
		
		circularBuffer<long int> getTimestampPointer() {
			return timestampPointer;
		}
		
		void addQuery(Query &query) {
			queries.push_back(&query);
		}
		
		void addTimeQuery(TimeQuery query) {
			timeQueries.push_back(query);
		}
		
		void addCountQuery(CountQuery query) {
			countQueries.push_back(query);
		}

		
		
		void copyElements (int deviceSpan, int hostSpan, int copySize) {
			hipMemcpy(deviceSpan + devicePointer.rdfStore.pointer, &rdfBuffer[0] + hostSpan, copySize * sizeof(tripleContainer), hipMemcpyHostToDevice); 
			hipMemcpy(deviceSpan + devicePointer.subject.pointer, &subjectBuffer[0] + hostSpan, copySize * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.predicate.pointer,&predicateBuffer[0] + hostSpan, copySize * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.object.pointer, &objectBuffer[0] + hostSpan, copySize * sizeof(int), hipMemcpyHostToDevice);
		}

		
		void advanceDevicePointer() {
			int copySize = rdfBuffer.size();
			
			circularBuffer<tripleContainer> rdfBuff = devicePointer.rdfStore;

			int newEnd = (rdfBuff.end + copySize) % rdfBuff.size;

				 
			if (newEnd < rdfBuff.end) {
				int finalHalf = rdfBuff.size - rdfBuff.end;
				copyElements(devicePointer.rdfStore.end, 0, finalHalf);			
	
				int firstHalf = copySize - finalHalf;
				copyElements(0, finalHalf, firstHalf);			
			} else {
				copyElements(devicePointer.rdfStore.end, 0, copySize);	
			}
			


			devicePointer.setEnd(newEnd);

			rdfBuffer.clear();
			subjectBuffer.clear();
			predicateBuffer.clear();
			objectBuffer.clear();
		}
		
		void checkStep() {	
			for (auto &query : queries)  {
				query->advancePointer(1);
				if (query->isReady()) {
					for (int i = timestampPointer.end -10 ; i <timestampPointer.end;  i++) {
				
				
				}
					int copySize = rdfBuffer.size();
					advanceDevicePointer();				
					query->launch();
				}
			}
		}
		
		void start() {
			
			int counter = 0;

			struct timeval startingTs;
			gettimeofday(&startingTs, NULL);
			long int ts = startingTs.tv_sec * 1000000 + startingTs.tv_usec;

			for (auto &query : queries) {
				query->setStartingTimestamp(ts);
			}
			
			std::cout << "Initial ts is " << ts << std::endl;
			
			for (int i =0; i <srcSize; i++) {
			

				tripleContainer currentTriple  = source[i];
				
				
				struct timeval tp;
				gettimeofday(&tp, NULL);
				long int ms = tp.tv_sec * 1000000 + tp.tv_usec;
				
				timestampPointer.pointer[timestampPointer.end] = ms;
				timestampPointer.end = (timestampPointer.end + 1) % timestampPointer.size;
				timestampPointer.begin = timestampPointer.end;
				
				rdfBuffer.push_back(currentTriple);
				subjectBuffer.push_back(currentTriple.subject);
				predicateBuffer.push_back(currentTriple.predicate);
				objectBuffer.push_back(currentTriple.object);
				
				struct timeval begin;
				gettimeofday(&begin, NULL);
				
				checkStep();

				
				
				usleep(spanTime);
			}
			
			
			//***** REMOVE IN FINAL CODE: ONLY FOR TEST *****
			int copySize = rdfBuffer.size();
			
			if (copySize != 0 ) {
				advanceDevicePointer();
					 
				for (auto &query : queries)  {
					query->launch();
				}
							
	
			}
			

			//***** END REMOVE PART-*****			
		}



};


__global__ void unarySelect (circularBuffer<int> src, int* value, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (abs(src.end - src.begin +  src.size) % src.size) ) {
		return;
	}	

	int newIndex = (src.begin + index) % src.size;
	
	if (src.pointer[newIndex] == (*value)) {
		int add = atomicAdd(size, 1);
		dest[add] = store.pointer[newIndex];
	}
}

__global__ void binarySelect (circularBuffer<int> src1, circularBuffer<int> src2, int* value1, int* value2, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (abs(src1.end - src1.begin + src1.size) % src1.size) ) {
		return;
	}		

	int newIndex = (src1.begin + index) % src1.size;
	if ((src1.pointer[newIndex] == (*value1)) && (src2.pointer[newIndex] == (*value2))) {
		int add = atomicAdd(size, 1);
		dest[add] = store.pointer[newIndex];
	}
}



/*
* Make multiple select query, with specified comparison condition,
* on a triple store. Both queries and the store are supposed to 
* be already on the device. 
* 
* @param d_selectQueries : the array in which are saved the select values
* @param d_storePointer : pointer on the device to the triple store
* @param storeSize : size of the triple store
* @param comparatorMask : array of triple of comparator that are applied to the queries
*			must be of the size of the d_selectQueries
* @return a vector of type mem_t in which are saved the query results.
*/
std::vector<mem_t<tripleContainer>*> rdfSelect(const std::vector<tripleContainer*> d_selectQueries, 
		deviceCircularBuffer d_pointer,
		const int storeSize, 
		std::vector<int>  arrs) 
{
	standard_context_t context;
	//Initialize elements
	int querySize =  d_selectQueries.size();
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	int* currentSize;
	hipMalloc(&currentSize, sizeof(int));
	int* zero = (int*) malloc(sizeof(int));
	*zero = 0;

	int* finalResultSize  = (int*) malloc(sizeof(int));

	//Cycling on all the queries
	for (int i = 0; i < querySize; i++) {
		//Save variable to pass to the lambda operator
		tripleContainer* currentPointer = d_selectQueries[i];
		
		mem_t<tripleContainer>* currentResult = new mem_t<tripleContainer>(storeSize, context);

		int gridSize = 300;
	        int blockSize = (storeSize / gridSize) + 1;
		hipMemcpy(currentSize, zero, sizeof(int), hipMemcpyHostToDevice);
			
		switch(arrs[i]) {

			case(0): {
				int* value = &(currentPointer->subject);

				unarySelect<<<gridSize,blockSize>>>(d_pointer.subject, value, currentResult->data(), d_pointer.rdfStore, currentSize);

				break;
			}

			case(1): {
				int* value = &(currentPointer->predicate);
			
				unarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, value, currentResult->data(), d_pointer.rdfStore, currentSize);

				break;
			}
						
			case(2): {
                                int* value = &(currentPointer->object);

                                unarySelect<<<gridSize,blockSize>>>(d_pointer.object, value, currentResult->data(), d_pointer.rdfStore, currentSize);
                                break;
			}
			
			case(3): {
				int* value1 = &(currentPointer->subject);
				int* value2 = &(currentPointer->predicate);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.predicate, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize);

				break;
			}

			case(4): {
				int* value1 = &(currentPointer->subject);
				int* value2 = &(currentPointer->object);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.object, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize);

				break;
			}

			case(5): {
				int* value1 = &(currentPointer->predicate);
				int* value2 = &(currentPointer->object);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, d_pointer.object, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize);

				break;
			}
						
			case(6): {
				hipMemcpy(currentResult->data(), d_pointer.rdfStore.pointer, storeSize * sizeof(tripleContainer), hipMemcpyDeviceToDevice);
				hipMemcpy(currentSize, &storeSize, sizeof(int), hipMemcpyHostToDevice);
                                break;
			}
			
			
			default: {
				printf("ERROR ERRROR ERROR ERROR ERROR ERROR ERROR");
			}


		}
				
                hipMemcpy(finalResultSize, currentSize, sizeof(int), hipMemcpyDeviceToHost);
		currentResult->setSize(*finalResultSize);
		finalResults.push_back(currentResult);
	}
	hipFree(currentSize);
	
	return finalResults;
}



//Sorter for sorting the triple due to theorder defined by the sortMask
 class TripleSorter {
	private:
		int sortMask[3];
	public:
		TripleSorter(JoinMask sortMask[3]) {
			this->sortMask[0] = static_cast<int> (sortMask[0]);
			this->sortMask[1] = static_cast<int> (sortMask[1]);
			this->sortMask[2] = static_cast<int> (sortMask[2]);
				
		}
		
		MGPU_DEVICE bool operator() (tripleContainer a, tripleContainer b) {
			int tripleA[3] = {a.subject, a.predicate, a.object};
			int tripleB[3] = {b.subject, b.predicate, b.object};
			
			if ((sortMask[0] != -1) && (tripleA[sortMask[0]] < tripleB[sortMask[0]])) {
				return true;
			}
			
			if ((sortMask[1] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] < tripleB[sortMask[1]])) {
				return true;
			}
			
			if ((sortMask[2] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] == tripleB[sortMask[1]]) && (tripleA[sortMask[2]] < tripleB[sortMask[2]])) {
				return true;
			}
			
			return false;
		}
};



struct mask_s {
	int subject;
	int predicate;
	int object;
};
 
__global__ void reorderTriple(tripleContainer* src, tripleContainer* dest, int maxSize, mask_s mask) {
		
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (destIndex  >= maxSize)  {
		return;
	}

	int triple[3] = {src[destIndex].subject, src[destIndex].predicate, src[destIndex].object};
 	tripleContainer destTriple = {triple[mask.subject], -1, -1};
 	
 	if (mask.predicate != -1) {
 		destTriple.predicate = triple[mask.predicate];
 	}
 	
 	
 	if (mask.object != -1) {
	 	destTriple.object = triple[mask.object];
 	}
 	
	dest[destIndex] = destTriple;
}


__global__ void indexCopy(tripleContainer* innerSrc, tripleContainer* innerDest, tripleContainer* outerSrc, tripleContainer* outerDest, int2* srcIndex, int maxSize) 
{
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (destIndex  >= maxSize)  {
		return;
	}
	
	//INVETERD INDEX DUE TO INVERTED JOIN PROBLEM (it should be inner = x, outer = y)
	int innerIndex = srcIndex[destIndex].y;
	int outerIndex = srcIndex[destIndex].x;
	
	innerDest[destIndex] = innerSrc[innerIndex];	
	outerDest[destIndex] = outerSrc[outerIndex];
}



std::vector<mem_t<tripleContainer>*> rdfJoin(tripleContainer* innerTable, int innerSize, tripleContainer* outerTable, int outerSize, JoinMask innerMask[3], JoinMask outerMask[3])
{
	standard_context_t context;
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	TripleSorter* innerSorter = new TripleSorter(innerMask);

	struct timeval beginCu, end;
	gettimeofday(&beginCu, NULL);
	mask_s mask;
	mask.subject = static_cast<int> (outerMask[0]);
	mask.predicate = static_cast<int> (outerMask[1]);
	mask.object = static_cast<int> (outerMask[2]);	
	int gridSize = 64;
	int blockSize = (outerSize/ gridSize) + 1;
	mem_t<tripleContainer>* tempOuter = new mem_t<tripleContainer>(outerSize, context);
	reorderTriple<<<gridSize, blockSize>>>(outerTable, tempOuter->data(), outerSize, mask);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	float cuTime = (end.tv_sec - beginCu.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginCu.tv_usec) / 1000 ;
	
	std::cout << "OVERHEAD TIME IS " << cuTime << std::endl;
	
	//Sort the two input array
	mergesort(innerTable, innerSize , *innerSorter, context);
	mergesort(tempOuter->data(), outerSize , *innerSorter, context);
	
	
	
	//BUG che mi costringe ad invertire inner con outer?
	mem_t<int2> joinResult = inner_join( innerTable, innerSize, tempOuter->data(), outerSize,  *innerSorter, context);
		
	mem_t<tripleContainer>* innerResults = new mem_t<tripleContainer>(joinResult.size(), context);
        mem_t<tripleContainer>* outerResults = new mem_t<tripleContainer>(joinResult.size(), context);
	
	//SETTARE DIVISIONE CORRETTA
	//BIsogna settare come comporatrsi quando il valore della join supera i 129k risultati
	gridSize = 64;
	blockSize = (joinResult.size() / gridSize) + 1; 
	indexCopy<<<gridSize, blockSize>>>(innerTable, innerResults->data(), outerTable, outerResults->data(), joinResult.data(), joinResult.size());

	finalResults.push_back(innerResults);
	finalResults.push_back(outerResults);

	return finalResults;
}




template<typename type_t, typename accuracy>
std::vector<accuracy> stats(std::vector<type_t> input) {
	int size = input.size();
	float mean = 0;
	float variance = 0;
	for (int i = 0; i < size; i++) {
		mean += (accuracy) input[i];
                variance += (accuracy)  (input[i] * input[i]);
        }
        mean = mean / ((accuracy) size);
        variance = variance / ((accuracy) size);
        variance = variance - (mean * mean);
        std::vector<accuracy> statistic;
	statistic.push_back(mean);
	statistic.push_back(variance);
	return statistic;
}

int main(int argc, char** argv) {
 
		using namespace std;
		struct timeval beginPr, beginCu, beginEx, end;
		gettimeofday(&beginPr, NULL);	
		hipDeviceReset();
		standard_context_t context;

                size_t BUFFER_SIZE = 400000;
		deviceCircularBuffer windowPointer;


		std::vector<float> timeCuVector;                
		std::vector<float> timeExVector;


		ifstream rdfStoreFile (argv[1]);
		string strInput;

		int fileLength = 0;	 
		while (std::getline(rdfStoreFile, strInput)) {
			++fileLength;
		}
	
		rdfStoreFile.clear();
		rdfStoreFile.seekg(0, ios::beg);

                size_t rdfSize = fileLength  * sizeof(tripleContainer);
                tripleContainer* h_rdfStore = (tripleContainer*) malloc(rdfSize);


		TripleGenerator manager(h_rdfStore, fileLength, 1, BUFFER_SIZE);

                //read store from rdfStore
                for (int i = 0; i <fileLength; i++) {
			getline(rdfStoreFile,strInput);
                        std::vector<string> triple;
                        separateWords(strInput, triple, ' ');
			
			h_rdfStore[i].subject = atoi(triple[0].c_str());
                        h_rdfStore[i].predicate = atoi(triple[1].c_str());
                        h_rdfStore[i].object = atoi(triple[2].c_str());

                }

                rdfStoreFile.close();

                int N_CYCLE = 1;
		for (int i = 0; i < N_CYCLE; i++) {
			gettimeofday(&beginCu, NULL);
			
			hipMalloc(&windowPointer.rdfStore.pointer, BUFFER_SIZE * sizeof(tripleContainer));
			hipMalloc(&windowPointer.subject.pointer, BUFFER_SIZE * sizeof(int));
                        hipMalloc(&windowPointer.predicate.pointer, BUFFER_SIZE * sizeof(int));
                        hipMalloc(&windowPointer.object.pointer, BUFFER_SIZE * sizeof(int));
					
			int begin = 0;
			windowPointer.setBegin(begin);
			windowPointer.setEnd(begin);
			windowPointer.setSize(BUFFER_SIZE);
			
			manager.setDevicePointer(windowPointer);
			
		        //set Queries (select that will be joined)
		        tripleContainer h_queryVector1 =  {-1, -1 , 99};
		        tripleContainer h_queryVector2 =	{0, -1, -1}; 

			cout << "query is: first obj: " << h_queryVector1.object << "; secons subj: " << h_queryVector2.subject << endl;

		        mem_t<tripleContainer> d_queryVector1(1, context);
			hipMemcpy(d_queryVector1.data(), &h_queryVector1, sizeof(tripleContainer), hipMemcpyHostToDevice);
		
		        mem_t<tripleContainer> d_queryVector2(1, context);
			hipMemcpy(d_queryVector2.data(), &h_queryVector2, sizeof(tripleContainer), hipMemcpyHostToDevice);
	
			//set select mask operation
			std::vector<tripleContainer*> selectQuery;
			selectQuery.push_back(d_queryVector1.data());
			selectQuery.push_back(d_queryVector2.data());

			SelectArr arr1 = SelectArr::SPO;
			SelectArr arr2 = SelectArr::SPO;
		
			//set Join mask
			JoinMask innerMask[3];
			innerMask[0] = JoinMask::PRE;
			innerMask[1] = JoinMask::NJ;
			innerMask[2] = JoinMask::NJ;
			
			JoinMask outerMask[3];
			outerMask[0] = JoinMask::PRE;
			outerMask[1] = JoinMask::NJ;
			outerMask[2] = JoinMask::NJ;

			//Creat operation object to pass to query manager
			SelectOperation  selectOp1(&d_queryVector1, arr1);
			SelectOperation  selectOp2(&d_queryVector2, arr2);
		
			JoinOperation  joinOp(selectOp1.getResultAddress(), selectOp2.getResultAddress(), innerMask, outerMask);
		
			std::vector<SelectOperation*> selectOperations;
			std::vector<JoinOperation*> joinOperations;
		
			selectOperations.push_back(&selectOp1);
			selectOperations.push_back(&selectOp2);
			joinOperations.push_back(&joinOp);
			
			int stepCount = 100000;

			TimeQuery count(selectOperations, joinOperations, windowPointer, manager.getTimestampPointer(), 10000, 10000);
			manager.addQuery(count);
			
		/*	CountQuery count(selectOperations, joinOperations, windowPointer, 150000);
			manager.addQuery(count);*/
			
			gettimeofday(&beginEx, NULL);	
			
			manager.start();

			
			/*//Retrive results from memory
			std::vector<tripleContainer> finalInnerResults = from_mem(*joinOp.getInnerResult());
			std::vector<tripleContainer> finalOuterResults = from_mem(*joinOp.getOuterResult());*/
			
			hipDeviceSynchronize();
			gettimeofday(&end, NULL);

                      /*  std::vector<tripleContainer> selectResults = from_mem(*selectOp1.getResult());
                        std::vector<tripleContainer> selectResults2 = from_mem(*selectOp2.getResult());*/

			float exTime = (end.tv_sec - beginEx.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginEx.tv_usec) / 1000 ;
			float prTime = (end.tv_sec - beginPr.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginPr.tv_usec) / 1000 ;
			float cuTime = (end.tv_sec - beginCu.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginCu.tv_usec) / 1000 ;
			
			timeCuVector.push_back(cuTime);
			timeExVector.push_back(exTime);
			/*
			//Print Results
			cout << "first select result" << endl;
			for (int i = 0; i < selectResults.size(); i++) {
				cout << selectResults[i].subject << " " << selectResults[i].predicate << " "  << selectResults[i].object << endl; 
			}*/
			

			
			//Print current cycle results
			/*cout <<"first Select Size " << selectResults.size() << endl;
			cout << "second Select Size " << selectResults2.size() << endl;
			cout << "join Size " << finalOuterResults.size() << endl;*/
						
			cout << "Total time: " << prTime << endl;
			cout << "Cuda time: " << cuTime << endl;
			cout << "Execution time: " << exTime << endl;					
			cout << "" << endl;


			hipFree(windowPointer.subject.pointer);
			hipFree(windowPointer.predicate.pointer);
			hipFree(windowPointer.object.pointer);
			hipFree(windowPointer.rdfStore.pointer);
		}

/*	
		std::vector<float> statistics;
		
		statistics = stats<float, float>(timeCuVector);	
                cout << "mean cuda time " << statistics[0] << endl;
                cout << "variance cuda time " << statistics[1] << endl;

                statistics = stats<float, float>(timeExVector);
                cout << "mean ex time " << statistics[0] << endl;
                cout << "variance ex time " << statistics[1] << endl;
*/

		cout << "FINAL VALUE IS " << TEST_VALUE[0] << std::endl;
		cout << "FINAL VALUE IS " << TEST_VALUE[1] << std::endl;
		
                return 0;
}
