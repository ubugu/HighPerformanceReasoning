#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <unistd.h>
#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_join.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <sys/time.h>
#include <sparsehash/dense_hash_map>

using namespace mgpu;
using google::dense_hash_map;

//TODO implementare la projection su gpu

//TODO 
//VARIABILI PER TESTING, DA RIMUOVERE DAL CODICE FINALE
int VALUE = 0;
std::vector<float> timeCuVector;                
std::vector<long int> timeExVector;
bool isLaunched = false;
//** END TESTING ***//


//struct to contains a single triple with int type.
struct tripleContainer {
        size_t subject;
        size_t predicate;
        size_t object;

	void print() {
		std::cout << subject << " " << predicate << " " << object << std::endl;
	}
};

//Struct for circular buffer
template<typename type_t>
struct circularBuffer {
	type_t* pointer;
	int begin;
	int end;
	int size;
	
	circularBuffer() : pointer(0), begin(0), end(0), size(0) {}
	
	int getLength() {
		return (abs(end - begin + size) % size);
	}
};

//Struct for containing the pointer to an rdf store (divided into subject predicate and object) 
template<typename rdf_t, typename arr_t>
struct triplePointer {
	rdf_t rdfStore;
	arr_t subject;
	arr_t predicate;
	arr_t object;
};


struct Binding {
	size_t* pointer;
	int width;
	int height;
	std::vector<std::string> header;
	
	Binding() {
	
	}
	
	Binding(int width, int height) {
		hipMalloc(&pointer, width * height *  sizeof(size_t));
		this->width = width;
		this->height = height;
	}
};


/*
* Specific implementation of triplePointer for ciruclar buffer.
* Offers methods for managing the attributes of the class.
*/
struct deviceCircularBuffer : triplePointer<circularBuffer<tripleContainer>, circularBuffer<size_t>> {
	void setValues(int begin, int end, int size) {
		setBegin(begin);
		setEnd(end);
		setSize(size);
	}	

	void setBegin(int begin) {
		rdfStore.begin = begin;
		subject.begin = begin;
		predicate.begin = begin;
		object.begin = begin;
	}
	
	void setEnd(int end) {
		rdfStore.end = end;
		subject.end = end;
		predicate.end = end;
		object.end = end;
	}
	
	void setSize(int size) {
		rdfStore.size = size;
		subject.size = size;
		predicate.size = size;
		object.size = size;
	}
	
	void advanceBegin(int step){
		int newBegin = (rdfStore.begin + step) % rdfStore.size;
		setBegin(newBegin);
	}
				
};

int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}

/*
* Join enum to define order and which element to join
* NJ indicates a non-join value, so it is ignored during join and sorting
* So that it improves performance avoiding uneecessary conditional expression
*/
enum class JoinMask {NJ = -1, SBJ = 0, PRE = 1, OBJ = 2};


//Section for defining operation classes
class JoinOperation 
{	
	//TODO Modificare classe in modo che permetta la join di join
	private:
		Binding** innerTable;
		Binding** outerTable;
		Binding* result = 0;
		
		std::vector<std::string> joinMask;

	public:
		JoinOperation(Binding** innerTable, Binding** outerTable, std::string joinMask) {
			this->innerTable = innerTable;
			this->outerTable = outerTable;
			separateWords(joinMask, this->joinMask, ' ');
		};
			
		Binding* getInnerTable() {
			return *this->innerTable;
		};
		
		Binding* getOuterTable() {
			return *this->outerTable;
		};
		
		std::vector<std::string> getJoinMask() {
			return this->joinMask;
		};
		

		Binding* getResult() {
			return this->result;
		};
		
		void setResult(Binding* result) {
			this->result = result;
		};
		

		
		
};

enum class SelectArr { S = 0, P = 1, O = 2, SP = 3, SO = 4, PO = 5, SPO = 6};




__global__ void unarySelect (circularBuffer<tripleContainer> src, int target, int first, int second, size_t* value, size_t* dest, int width, int* size) {

			int index = blockIdx.x * blockDim.x + threadIdx.x;

			if (index >= (abs(src.end - src.begin +  src.size) % src.size) ) {
				return;
			}	

			int newIndex = (src.begin + index) % src.size;
	
			size_t temp[3] = {src.pointer[newIndex].subject, src.pointer[newIndex].predicate, src.pointer[newIndex].object};
	
			if (temp[target] == (*value)) {
				int add = atomicAdd(size, 1);
				size_t* dest_p = (size_t*) (dest + add * width) ;



				*dest_p = temp[first];
				*(dest_p + 1) = temp[second];
		
			}
		}


		__global__ void binarySelect (circularBuffer<size_t> src1, circularBuffer<size_t> src2, size_t* value1, size_t* value2, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size) {

			int index = blockIdx.x * blockDim.x + threadIdx.x;
		
			if (index >= (abs(src1.end - src1.begin + src1.size) % src1.size) ) {
				return;
			}		

			int newIndex = (src1.begin + index) % src1.size;
			if ((src1.pointer[newIndex] == (*value1)) && (src2.pointer[newIndex] == (*value2))) {
				int add = atomicAdd(size, 1);
				dest[add] = store.pointer[newIndex];
		
			}
		}




class SelectOperation 
{
	private:
		mem_t<tripleContainer>* query;
		Binding* result;
		int arr;
		std::vector<std::string> variables;

	public:
		SelectOperation(mem_t<tripleContainer>* query, SelectArr arr, std::string variable) {
			this->query = query;	
			this->arr = static_cast<int> (arr);
			separateWords(variable, variables, ' ');
		};

		int getArr() {
			return this-> arr;
		}
			
		mem_t<tripleContainer>* getQuery() {
			return this->query;
		};
		                                                                            
		
		void setResult(Binding* result) {
			this->result = result;
		};
		
		Binding* getResult() {
			return result;
		};
		
		std::vector<std::string> getVariables() {
			return variables;
		};
		
		Binding** getResultAddress() {
			return &result;
		}


		





		/*
		* Make multiple select query, with specified comparison condition,
		* on a triple store. Both queries and the store are supposed to 
		* be already on the device. 
		* 
		* @param d_selectQueries : the array in which are saved the select values
		* @param d_storePointer : pointer on the device to the triple store
		* @param storeSize : size of the triple store
		* @return a vector of type mem_t in which are saved the query results.
		*/
		void rdfSelect(deviceCircularBuffer d_pointer, const int storeSize) 
		{	
			//Initialize elements	
			int* d_resultSize;
			hipMalloc(&d_resultSize, sizeof(int));
			int h_resultSize  = 0;

			hipMemcpy(d_resultSize, &h_resultSize, sizeof(int), hipMemcpyHostToDevice);
	
			//INSERIRE DIVISIONE CORRETTA
			int gridSize = 300;
			int blockSize = (storeSize / gridSize) + 1;
			tripleContainer* query = this->query->data();
		
			result = new Binding(2, storeSize);
						
			switch(arr) {

				case(0): {
					size_t* value = &(query->subject);
					unarySelect<<<gridSize,blockSize>>>(d_pointer.rdfStore, 0, 1, 2, value, result->pointer, result->width, d_resultSize);
					break;
				}

				case(1): {
					size_t* value = &(query->predicate);
					unarySelect<<<gridSize,blockSize>>>(d_pointer.rdfStore,  1, 0, 2, value, result->pointer, result->width, d_resultSize);
					break;
				}
					
				case(2): {
				
			                size_t* value = &(query->object);
	
			                unarySelect<<<gridSize,blockSize>>>(d_pointer.rdfStore,  2, 0, 1, value, result->pointer, result->width, d_resultSize);
			             
			                break;
				}
		
		/*		case(3): {
					size_t* value1 = &(query->subject);
					size_t* value2 = &(query->predicate);
					binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.predicate, value1, value2, result->data(), d_pointer.rdfStore, d_resultSize);
					break;
				}

				case(4): {
					size_t* value1 = &(query->subject);
					size_t* value2 = &(query->object);
					binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.object, value1, value2, result->data(), d_pointer.rdfStore, d_resultSize);
					break;
				}

				case(5): {
					size_t* value1 = &(query->predicate);
					size_t* value2 = &(query->object);
					binarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, d_pointer.object, value1, value2, result->data(), d_pointer.rdfStore, d_resultSize);
					break;
				}
					
				case(6): {
					hipMemcpy(result->data(), d_pointer.rdfStore.pointer, storeSize * sizeof(tripleContainer), hipMemcpyDeviceToDevice);
					hipMemcpy(d_resultSize, &storeSize, sizeof(int), hipMemcpyHostToDevice);
			                break;
				}*/
		
			}
			hipDeviceSynchronize();
	
	
			hipMemcpy(&h_resultSize, d_resultSize, sizeof(int), hipMemcpyDeviceToHost);

			result->height  =  h_resultSize;		

			hipFree(d_resultSize);

		}




	
};


//Sorter for sorting the triple due to theorder defined by the sortMask
 class TripleSorter {
	private:
		int sortMask[3];
	public:
		TripleSorter(JoinMask sortMask[3]) {
			this->sortMask[0] = static_cast<int> (sortMask[0]);
			this->sortMask[1] = static_cast<int> (sortMask[1]);
			this->sortMask[2] = static_cast<int> (sortMask[2]);
				
		}
		
		MGPU_DEVICE bool operator() (tripleContainer a, tripleContainer b) {
			size_t tripleA[3] = {a.subject, a.predicate, a.object};
			size_t tripleB[3] = {b.subject, b.predicate, b.object};

							
			if ((sortMask[0] != -1) && (tripleA[sortMask[0]] < tripleB[sortMask[0]])) {
				return true;
			}
			
			if ((sortMask[1] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] < tripleB[sortMask[1]])) {
				return true;
			}
			
			if ((sortMask[2] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] == tripleB[sortMask[1]]) && (tripleA[sortMask[2]] < tripleB[sortMask[2]])) {
				return true;
			}
			
			return false;
		}
};






//Sorter for sorting the triple due to theorder defined by the sortMask
 class Sorter {
	private:
		int sortMask[3];
	public:
		Sorter(int sortMask[3]) {
			this->sortMask[0] =  (sortMask[0]);
			this->sortMask[1] =  (sortMask[1]);
			this->sortMask[2] =  (sortMask[2]);
				
		}
		
		
		MGPU_DEVICE bool operator() (size_t a, size_t b) {
		
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			        
			printf("pointers are %p , %p \n", &a, &b);
		
			/*size_t tripleA[3] = {a.subject, a.predicate, a.object};
			size_t tripleB[3] = {b.subject, b.predicate, b.object};*/

				/*			
			if ((sortMask[0] != -1) && (tripleA[sortMask[0]] < tripleB[sortMask[0]])) {
				return true;
			}
			
			if ((sortMask[1] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] < tripleB[sortMask[1]])) {
				return true;
			}
			
			if ((sortMask[2] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] == tripleB[sortMask[1]]) && (tripleA[sortMask[2]] < tripleB[sortMask[2]])) {
				return true;
			}*/
			
			return false;
		}
};





struct mask_t {
	int subject;
	int predicate;
	int object;
};
 

__global__ void reorderTriple(tripleContainer* src, tripleContainer* dest, int maxSize, mask_t mask) {
		
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (destIndex  >= maxSize)  {
		return;
	}

	size_t triple[3] = {src[destIndex].subject, src[destIndex].predicate, src[destIndex].object};

	dest[destIndex] = {triple[mask.subject], triple[mask.predicate], triple[mask.object]};
}


__global__ void indexCopy(tripleContainer* innerSrc, tripleContainer* innerDest, tripleContainer* outerSrc, tripleContainer* outerDest, int2* srcIndex, int maxSize) 
{
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (destIndex  >= maxSize)  {
		return;
	}
	
	int innerIndex = srcIndex[destIndex].x;
	int outerIndex = srcIndex[destIndex].y;
	
	innerDest[destIndex] = innerSrc[innerIndex];	
	outerDest[destIndex] = outerSrc[outerIndex];
}





std::vector<mem_t<tripleContainer>*> rdfJoin(Binding* innerTable, Binding* outerTable, std::vector<std::string> joinMask)
{
	//TODO Migliorare la join nel reordering delle triple
	standard_context_t context;
	std::vector<mem_t<tripleContainer>*> finalResults;
	
		
	
/*
	


	mask_t mask;
	mask.subject = static_cast<int> (outerMask[0]);
	mask.predicate = static_cast<int> (outerMask[1]);

	mask.object = static_cast<int> (outerMask[2]);	
	
	//TODO SETTARE DIVISIONE
	int gridSize = 124;
	int blockSize = (outerSize/ gridSize) + 1;
	mem_t<tripleContainer>* tempOuter = new mem_t<tripleContainer>(outerSize, context);

	reorderTriple<<<gridSize, blockSize>>>(outerTable, tempOuter->data(), outerSize, mask);


	
	std::cout << "LAUNCH MEGER " << std::endl;

	//Sort the two input array
	mergesort<launch_params_t<128, 2>>(d_iter, innerTable->height, *innerSorter, context);
	exit(1);
//	mergesort<launch_params_t<128, 2>>(outerTable->pointer, outerSize , *innerSorter, context);
/*	
	mem_t<int2> joinResult = inner_join<launch_params_t<128,2>>( innerTable, innerSize, tempOuter->data(), outerSize,  *innerSorter, context);
		
	std::cout << "JOIN RESULT SIZE IS " << joinResult.size() << std::endl;
	
	mem_t<tripleContainer>* innerResults = new mem_t<tripleContainer>(joinResult.size(), context);
        mem_t<tripleContainer>* outerResults = new mem_t<tripleContainer>(joinResult.size(), context);
	
	//SETTARE DIVISIONE CORRETTA
	//TODO BIsogna settare come comporatrsi quando il valore della join supera i 129k risultati
	gridSize = 64;
	blockSize = (joinResult.size() / gridSize) + 1; 
	indexCopy<<<gridSize, blockSize>>>(innerTable, innerResults->data(), tempOuter->data(), outerResults->data(), joinResult.data(), joinResult.size());

	finalResults.push_back(innerResults);
	finalResults.push_back(outerResults);
	hipFree(tempOuter->data());
	free(tempOuter);
	*/
	return finalResults;
}


class Query {
	protected:
		std::vector<SelectOperation*> select;
		std::vector<JoinOperation*> join;
		deviceCircularBuffer windowPointer;

	public:
		Query(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join, deviceCircularBuffer rdfPointer) {
			this->join = join;
			this->select = select;
			this->windowPointer = rdfPointer;
		}

		virtual void setWindowEnd(int step) {
			windowPointer.setEnd(step);
		}
		
		/**
		* Function for managing query execution
		**/
		//TODO Verificare se si puo migliorare
		void startQuery() {
			int storeSize =  windowPointer.rdfStore.getLength();			
			
			for (auto op : select) {
				op->rdfSelect(windowPointer, storeSize);
			}

		
			/*
			
	
			for (int i = 0; i < join.size(); i++) {
				Binding* innerTable = join[i]->getInnerTable();
				Binding* outerTable = join[i]->getOuterTable();
				std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin(innerTable, outerTable, join[i]->getJoinMask());
			/*	join[i]->setInnerResult(joinResult[0]);
				join[i]->setOuterResult(joinResult[1]);			
			}*/
			
	
		}

		//TODO modificare quando si sapra come utilizzare i risultati
		void printResults(dense_hash_map<size_t, std::string> map) {

			int w = 0;
			for (auto op : select) {
				printf("W VALUE IS %i \n", w);
				if (w == 0) VALUE += op->getResult()->height;
				
				Binding* d_result = op->getResult();
				
				size_t* final_binding = (size_t*) malloc(d_result->height * d_result->width * sizeof(size_t));
				hipMemcpy(final_binding, d_result->pointer, d_result->width * sizeof(size_t) * d_result->height, hipMemcpyDeviceToHost);
				std::cout << "size is " << d_result->height << std::endl;
				std::cout << "width is " << d_result->width << std::endl;
				
				for (int z = 0; z < d_result->header.size(); z++) {
					std::cout << "header are " << d_result->header[z] << std::endl;
				}
				for (int i =0; i < d_result->height; i++) {
					//for (int k = 0; k < d_result->width; k++) {
						//size_t current = final_binding[i + k];
						std::cout << "result is " << map[ final_binding[i]] << " " <<  map[final_binding[i + 1]] << std::endl;
					//}
					
				}	
			
				w++;
				hipFree(d_result->pointer);
				
				

			}
	

		/*	for (auto op : join) {
				std::cout << "join result is " << op->getInnerResult()->size() << std::endl;

				std::vector<tripleContainer> innerRes = from_mem(*op->getInnerResult());
				std::vector<const char*> innerHash;

				for (int i =0; i < innerRes.size(); i++) {
					innerHash.push_back( map[innerRes[i].subject]);
                                        innerHash.push_back( map[innerRes[i].predicate]);
                                        innerHash.push_back( map[innerRes[i].object]);

				}
				
				std::vector<tripleContainer> outerRes = from_mem(*op->getOuterResult());
				std::vector<const char*> outerHash;
				for (int i =0; i< outerRes.size(); i++) {
					outerHash.push_back( map[outerRes[i].subject]);
                                        outerHash.push_back( map[outerRes[i].predicate]);
                                        outerHash.push_back( map[outerRes[i].object]);

				}

				VALUE += op->getInnerResult()->size();
				hipFree(op->getInnerResult()->data());
				hipFree(op->getOuterResult()->data());
			}*/
					
		}
		
	
};


class CountQuery : public Query {
	private:
		int count;
		int currentCount;

	public:
		CountQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer,
			int count) : Query(select, join, rdfPointer) {

				this->count = count;
				this->currentCount = 0;
		}
		
		void incrementCount() {
			this->currentCount++;
		}
		
		bool isReady() {
			return (currentCount == count);
		}
		
		void launch() {
			startQuery();
			windowPointer.advanceBegin(count);
			currentCount = 0;
		}
		
		~CountQuery() {}
};

class TimeQuery : public Query {
	private:
		circularBuffer<long int> timestampPointer;
		long int stepTime;
		long int windowTime;
		long int lastTimestamp;
		
	public:
		TimeQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer, circularBuffer<long int> timestampPointer,
			int windowTime, int stepTime) : Query(select, join, rdfPointer) {
				this->stepTime = stepTime;
				this->windowTime = windowTime;
				this->lastTimestamp = 0;
				this->timestampPointer = timestampPointer;
		}
		
		void setWindowEnd(int step)  {
			Query::setWindowEnd(step);
			timestampPointer.end = step;
		}

		bool isReady(long int newTimestamp) {
			return (lastTimestamp + windowTime < newTimestamp);
		}

		void setStartingTimestamp(long int timestamp) {
			this->lastTimestamp = timestamp;
		}

		void launch() {	
			//Update new starting value of buffer
			int newBegin = 0;
			for(int i = timestampPointer.begin; i  != timestampPointer.end; i = (i + 1) % timestampPointer.size) {	
				if (timestampPointer.pointer[i] > lastTimestamp) {
					newBegin = i;
					break;
				}				
			}				
			windowPointer.setBegin(newBegin);
			timestampPointer.begin = newBegin;
			
			//Lancuh query and print results
			startQuery();
	
			//Update window timestamp value
			lastTimestamp += stepTime;
		}

		~TimeQuery() {}
};



template <std::size_t FnvPrime, std::size_t OffsetBasis>
struct basic_fnv_1
{
    std::size_t operator()(std::string const& text) const
    {
        std::size_t hash = OffsetBasis;
         for(std::string::const_iterator it = text.begin(), end = text.end();
                 it != end; ++it)
         {
             hash *= FnvPrime;
             hash ^= *it;
         }
         return hash;

     }
};

dense_hash_map<size_t, std::string> map;


class QueryManager {
	private:
		int spanTime;
		std::string* source;
		int srcSize;
		
		std::vector<tripleContainer> rdfBuffer;
		std::vector<size_t> subjectBuffer;
		std::vector<size_t> predicateBuffer;
		std::vector<size_t> objectBuffer;

		std::vector<TimeQuery> timeQueries;
		std::vector<CountQuery> countQueries;
		
		circularBuffer<long int> timestampPointer;
		deviceCircularBuffer devicePointer;
    //            dense_hash_map<size_t, std::string> map;


	public:
		QueryManager(std::string* source, int srcSize, int spanTime,int buffSize)   {
			this->spanTime = spanTime;
			this->srcSize = srcSize;
			this->source = source;
			
			long int* timestamp = (long int*) malloc(buffSize * sizeof(long int));
			timestampPointer.pointer = timestamp;
			timestampPointer.size = buffSize;
		}
		
		void setDevicePointer(deviceCircularBuffer devicePointer) {
			this->devicePointer = devicePointer;
		}
		
		circularBuffer<long int> getTimestampPointer() {
			return timestampPointer;
		}
				
		void addTimeQuery(TimeQuery query) {
			timeQueries.push_back(query);
		}
		
		void addCountQuery(CountQuery query) {
			countQueries.push_back(query);
		}

				
		void copyElements (int deviceSpan, int hostSpan, int copySize) {
			hipMemcpy(deviceSpan + devicePointer.rdfStore.pointer, &rdfBuffer[0] + hostSpan, copySize * sizeof(tripleContainer), hipMemcpyHostToDevice); 
			hipMemcpy(deviceSpan + devicePointer.subject.pointer, &subjectBuffer[0] + hostSpan, copySize * sizeof(size_t), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.predicate.pointer,&predicateBuffer[0] + hostSpan, copySize * sizeof(size_t), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.object.pointer, &objectBuffer[0] + hostSpan, copySize * sizeof(size_t), hipMemcpyHostToDevice);
		}

		
		void advanceDevicePointer() {
			int copySize = rdfBuffer.size();
			
			circularBuffer<tripleContainer> rdfBuff = devicePointer.rdfStore;

			int newEnd = (rdfBuff.end + copySize) % rdfBuff.size;
	 
			if (newEnd < rdfBuff.end) {
				int finalHalf = rdfBuff.size - rdfBuff.end;
				copyElements(devicePointer.rdfStore.end, 0, finalHalf);			
	
				int firstHalf = copySize - finalHalf;
				copyElements(0, finalHalf, firstHalf);			
			} else {
				copyElements(devicePointer.rdfStore.end, 0, copySize);	
			}

			devicePointer.setEnd(newEnd);

			rdfBuffer.clear();
			subjectBuffer.clear();
			predicateBuffer.clear();
			objectBuffer.clear();
		}
		
		void checkStep() {	
			for (auto &query : countQueries)  {
				query.incrementCount();
				if (query.isReady()) {
					advanceDevicePointer();
					query.setWindowEnd(devicePointer.rdfStore.end);			
					query.launch();
					query.printResults(map);
				}
			}
			
			for (auto &query : timeQueries) {
				if (query.isReady(timestampPointer.pointer[timestampPointer.end - 1])) {
					advanceDevicePointer();
					query.setWindowEnd(devicePointer.rdfStore.end - 1);		
					query.launch();
					query.printResults(map);
					query.setWindowEnd(1);
				}				
			}
		}
		
		void start() {
			struct timeval startingTs;
			gettimeofday(&startingTs, NULL);
			long int ts = startingTs.tv_sec * 1000000 + startingTs.tv_usec;

			for (auto &query : timeQueries) {
				query.setStartingTimestamp(ts);
			}
			
			usleep(1);

			basic_fnv_1< 1099511628211u, 14695981039346656037u> h_func;

			map.set_empty_key(NULL);       
		

			for (int i =0; i <srcSize; i++) {

				
				tripleContainer currentTriple;
 
                                std::vector<std::string> triple;
                                separateWords(source[i], triple, ' ');
			
			        currentTriple.subject = h_func(triple[0]);
                                currentTriple.predicate = h_func(triple[1]);
                                currentTriple.object = h_func(triple[2]);

				map[currentTriple.subject] = triple[0];
                                map[currentTriple.predicate] = triple[1];
                                map[currentTriple.object] = triple[2] ;

				struct timeval tp;
				gettimeofday(&tp, NULL);
				long int ms = tp.tv_sec * 1000000 + tp.tv_usec;


				timestampPointer.pointer[timestampPointer.end] = ms;
				timestampPointer.end = (timestampPointer.end + 1) % timestampPointer.size;
				timestampPointer.begin = timestampPointer.end;
				
				rdfBuffer.push_back(currentTriple);
				subjectBuffer.push_back(currentTriple.subject);
				predicateBuffer.push_back(currentTriple.predicate);
				objectBuffer.push_back(currentTriple.object);
							
				checkStep();
			//	usleep(spanTime);

			}
			
			//TODO vedere se occorre tenere o no quest'ultima parte
			//***** REMOVE IN FINAL CODE: ONLY FOR TEST (FORSE) *****
			//DA OTTIMIZZARE POICHE VIENE LANCIATO ANCHE QUANDO NON SERVE!!
			advanceDevicePointer();	 
			for (auto &query : timeQueries)  {
				query.setWindowEnd(devicePointer.rdfStore.end);
				query.launch();
				query.printResults(map);
			}
			
			for (auto &query :countQueries) {
				query.setWindowEnd(devicePointer.rdfStore.end);
				query.launch();
				query.printResults(map);
			}
			//***** END REMOVE PART-*****			
		}



};





template<typename type_t, typename accuracy>
std::vector<accuracy> stats(std::vector<type_t> input) {
	int size = input.size();
	float mean = 0;
	float variance = 0;
	for (int i = 0; i < size; i++) {
		mean += (accuracy) input[i];
                variance += (accuracy)  (input[i] * input[i]);
        }
        mean = mean / ((accuracy) size);
        variance = variance / ((accuracy) size);
        variance = variance - (mean * mean);
        std::vector<accuracy> statistic;
	statistic.push_back(mean);
	statistic.push_back(variance);
	return statistic;
}

int main(int argc, char** argv) {
 
		using namespace std;
		
		struct timeval beginPr, beginCu, beginEx, end;
		gettimeofday(&beginPr, NULL);	
		hipDeviceReset();
		standard_context_t context;

                size_t BUFFER_SIZE = 400000;
		deviceCircularBuffer windowPointer;




		ifstream rdfStoreFile (argv[1]);
		string strInput;

		int fileLength = 0;	 
		while (std::getline(rdfStoreFile, strInput)) {
			++fileLength;
		}
	
		rdfStoreFile.clear();
		rdfStoreFile.seekg(0, ios::beg);

                size_t rdfSize = fileLength  * sizeof(std::string);
                std::string* h_rdfStore = (std::string*) malloc(rdfSize);


                //read store from rdfStore
                for (int i = 0; i <fileLength; i++) {
			getline(rdfStoreFile,strInput);
                        h_rdfStore[i]  = strInput;
                }

                rdfStoreFile.close();

                int N_CYCLE = 1;
		for (int i = 0; i < N_CYCLE; i++) {
			
			gettimeofday(&beginCu, NULL);
			QueryManager manager(h_rdfStore, fileLength, 1, BUFFER_SIZE);
			hipMalloc(&windowPointer.rdfStore.pointer, BUFFER_SIZE * sizeof(tripleContainer));
			hipMalloc(&windowPointer.subject.pointer, BUFFER_SIZE * sizeof(size_t));
                        hipMalloc(&windowPointer.predicate.pointer, BUFFER_SIZE * sizeof(size_t));
                        hipMalloc(&windowPointer.object.pointer, BUFFER_SIZE * sizeof(size_t));
					
			int begin = 0;
			windowPointer.setBegin(begin);
			windowPointer.setEnd(begin);
			windowPointer.setSize(BUFFER_SIZE);
			
			manager.setDevicePointer(windowPointer);
			basic_fnv_1< 1099511628211u, 14695981039346656037u> h_func;
		        //set Queries (select that will be joined)
		        tripleContainer h_queryVector1;
			h_queryVector1.subject = 0;
			h_queryVector1.predicate = 0;
			h_queryVector1.object =  h_func("<http://example.org/int/99>");
			
			
		        tripleContainer h_queryVector2;	
			h_queryVector2.subject = h_func("<http://example.org/int/0>");
			h_queryVector2.predicate = 0;
			h_queryVector2.object = 0;

			cout << "query is: first obj: " << h_queryVector1.object << "; secons subj: " << h_queryVector2.subject << endl;

		        mem_t<tripleContainer> d_queryVector1(1, context);
			hipMemcpy(d_queryVector1.data(), &h_queryVector1, sizeof(tripleContainer), hipMemcpyHostToDevice);
		
		        mem_t<tripleContainer> d_queryVector2(1, context);
			hipMemcpy(d_queryVector2.data(), &h_queryVector2, sizeof(tripleContainer), hipMemcpyHostToDevice);
	
			//set select mask operation
			std::vector<tripleContainer*> selectQuery;
			selectQuery.push_back(d_queryVector1.data());
			selectQuery.push_back(d_queryVector2.data());

			SelectArr arr1 = SelectArr::O;
			SelectArr arr2 = SelectArr::S;
		

			//Creat operation object to pass to query manager
			SelectOperation  selectOp1(&d_queryVector1, arr1, "?s ?p");
		//	SelectOperation  selectOp2(&d_queryVector2, arr2, "?p ?o");
		
		//	JoinOperation  joinOp(selectOp1.getResultAddress(), selectOp2.getResultAddress(), "?p");
		
			std::vector<SelectOperation*> selectOperations;
			std::vector<JoinOperation*> joinOperations;
		
			selectOperations.push_back(&selectOp1);
		//	selectOperations.push_back(&selectOp2);
		//	joinOperations.push_back(&joinOp);
			
			int stepCount = 100000;
			//std::cout << "starting tmsp " << manager.getTimestampPointer().begin << std::endl;
			
			
			
			/*TimeQuery count(selectOperations, joinOperations, windowPointer, manager.getTimestampPointer(), 5000, 5000);
			manager.addTimeQuery(count);


			TimeQuery count5(selectOperations, joinOperations, windowPointer, manager.getTimestampPointer(), 7000, 7000);
			manager.addTimeQuery(count5);*/
			
			CountQuery count2(selectOperations, joinOperations, windowPointer, 50000);
			manager.addCountQuery(count2);
			
			
			/*CountQuery count3(selectOperations, joinOperations, windowPointer, 50000);
			manager.addCountQuery(count3);
			
			CountQuery count4(selectOperations, joinOperations, windowPointer, 34652);
			manager.addCountQuery(count4);	*/		
			
			gettimeofday(&beginEx, NULL);	

			manager.start();


			
			hipDeviceSynchronize();
			gettimeofday(&end, NULL);


			float exTime = (end.tv_sec - beginEx.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginEx.tv_usec) / 1000 ;
			float prTime = (end.tv_sec - beginPr.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginPr.tv_usec) / 1000 ;
			float cuTime = (end.tv_sec - beginCu.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginCu.tv_usec) / 1000 ;
			

			
			timeCuVector.push_back(cuTime);

						
			cout << "Total time: " << prTime << endl;
			cout << "Cuda time: " << cuTime << endl;
			cout << "Execution time: " << exTime << endl;					
			cout << "" << endl;


			hipFree(windowPointer.subject.pointer);
			hipFree(windowPointer.predicate.pointer);
			hipFree(windowPointer.object.pointer);
			hipFree(windowPointer.rdfStore.pointer);
		}

	
		std::vector<float> statistics;
		
		statistics = stats<float, float>(timeCuVector);	
                cout << "mean cuda time " << statistics[0] << endl;
                cout << "variance cuda time " << statistics[1] << endl;

              /*  statistics = stats<long int, double>(timeExVector);
                cout << "mean ex time " << statistics[0] << endl;
                cout << "variance ex time " << statistics[1] << endl;*/


		cout << "FINAL VALUE IS " << VALUE << std::endl;;
		
		long int sum = 0;
		
		for (int i = 0; i < timeCuVector.size(); i++) {
			std::cout<< "time are " << timeCuVector[i] << std::endl;
		}
		

		
                return 0;
}
