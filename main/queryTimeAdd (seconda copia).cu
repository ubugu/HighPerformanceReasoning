#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <unistd.h>
#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_join.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <sys/time.h>

using namespace mgpu;

//struct to contains a single triple with int type.
struct tripleContainer {
        int subject;
        int predicate;
        int object;
};

template<typename type_t>
struct circularBuffer {
	type_t* pointer;
	int begin;
	int end;
	int size;
	
	circularBuffer() : pointer(0), begin(0), end(0), size(0) {}
	
	circularBuffer(int begin, int size, type_t* pointer) {
		this->begin = begin;
		this->end = begin;
		this->size = size;
		this->pointer = pointer;
	}

};

template<typename rdf_t, typename arr_t>
struct triplePointer {
	rdf_t rdfStore;
	arr_t subject;
	arr_t predicate;
	arr_t object;
};

struct deviceCircularBuffer : triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> {
	void setValues(int begin, int end, int size) {
		setBegin(begin);
		setEnd(end);
		setSize(size);
	}	

	void setBegin(int begin) {
		rdfStore.begin = begin;
		subject.begin = begin;
		predicate.begin = begin;
		object.begin = begin;
	}
	
	void setEnd(int end) {
		rdfStore.end = end;
		subject.end = end;
		predicate.end = end;
		object.end = end;
	}
	
	void setSize(int size) {
		rdfStore.size = size;
		subject.size = size;
		predicate.size = size;
		object.size = size;
	}
	
	void advanceBegin(int step){
		int newBegin = (rdfStore.begin + step) % rdfStore.size;
		setBegin(newBegin);
	}
	
	void advanceEnd(int step){
		int newEnd = (rdfStore.end + step) % rdfStore.size;
		setEnd(newEnd);
	}
	
				
};

int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}

/*
* Join enum to define order and which element to join
* NJ indicates a non-join value, so it is ignored during join and sorting
* So that it improves performance avoiding uneecessary conditional expression
*/
enum class JoinMask {NJ = -1, SBJ = 0, PRE = 1, OBJ = 2};


//Section for defining operation classes
class JoinOperation 
{	
	private:
		mem_t<tripleContainer>** innerTable;
		mem_t<tripleContainer>** outerTable;
		mem_t<tripleContainer>* innerResult = 0;
		mem_t<tripleContainer>* outerResult = 0;
		
		JoinMask innerMask[3];
		JoinMask outerMask[3];

	public:
		JoinOperation(mem_t<tripleContainer>** innerTable, mem_t<tripleContainer>** outerTable, JoinMask innerMask[3], JoinMask outerMask[3]) {
			this->innerTable = innerTable;
			this->outerTable = outerTable;
			std::copy(innerMask, innerMask + 3, this->innerMask);
			std::copy(outerMask, outerMask + 3, this->outerMask);
		};
			
		mem_t<tripleContainer>** getInnerTable() {
			return this->innerTable;
		};
		
		mem_t<tripleContainer>** getOuterTable() {
			return this->outerTable;
		};
		
		JoinMask* getInnerMask() {
			return this->innerMask;
		};
		
		JoinMask* getOuterMask() {
			return this->outerMask;
		};
		
		mem_t<tripleContainer>* getInnerResult() {
			return this->innerResult;
		};
		
		void setInnerResult(mem_t<tripleContainer>* result) {
			this->innerResult = result;
		};
		
		mem_t<tripleContainer>** getInnerResultAddress() {
			return &innerResult;
		}
		
		mem_t<tripleContainer>* getOuterResult() {
			return this->outerResult;
		};
		
		void setOuterResult(mem_t<tripleContainer>* result) {
			this->outerResult = result;
		};
		
		mem_t<tripleContainer>** getOuterResultAddress() {
			return &outerResult;
		}		
};

enum class SelectArr { S = 0, P = 1, O = 2, SP = 3, SO = 4, PO = 5, SPO = 6};

class SelectOperation 
{
	private:
		mem_t<tripleContainer>* query;
		mem_t<tripleContainer>* result = 0;
		int arr;

	public:
		SelectOperation(mem_t<tripleContainer>* query, SelectArr arr) {
			this->query = query;	
			this->arr = static_cast<int> (arr);
		};

		int getArr() {
			return this-> arr;
		}
			
		mem_t<tripleContainer>* getQuery() {
			return this->query;
		};
		                                                                            
		mem_t<tripleContainer>* getResult() {
			return this->result;
		};
		
		void setResult(mem_t<tripleContainer>* result) {
			this->result = result;
		};
		
		mem_t<tripleContainer>** getResultAddress() {
			return &result;
		}
};


class Query {
	public:
		std::vector<SelectOperation*> select;
		std::vector<JoinOperation*> join;
		deviceCircularBuffer windowPointer;

		Query(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join, deviceCircularBuffer rdfPointer) {
			this->join = join;
			this->select = select;
			this->windowPointer = rdfPointer;
		}

		std::vector<SelectOperation*> getSelect() {
			return select;
		}

		std::vector<JoinOperation*> getJoin() {
			return join;
		}

		void advancePointer(int step) {
			windowPointer.advanceEnd(step);
		}

		triplePointer<tripleContainer*, int*> getStorePointer() {
			triplePointer<tripleContainer*, int*> pointer;
			pointer.rdfStore = windowPointer.rdfStore.pointer;
			pointer.subject = windowPointer.subject.pointer;
			pointer.predicate = windowPointer.predicate.pointer;
			pointer.object = windowPointer.object.pointer;
			
			return pointer;
		}
		
		void printResults() {
			
			for (auto op : select) {
				std::vector<tripleContainer> selectResults = from_mem(*(op->getResult()));
				std::cout <<"selct size " << selectResults.size() << std::endl;
				
				hipFree(op->getResult()->data());
			}
			
			for (auto op : join) {
				hipFree(op->getInnerResult()->data());
				hipFree(op->getOuterResult()->data());
			}
					
		}
		
	
};

void queryManager(std::vector<SelectOperation*> selectOp, std::vector<JoinOperation*> joinOp, 
		deviceCircularBuffer d_pointer, 
		const int storeSize);

class CountQuery : public Query {
	private:
		int count;
		int currentCount;

	public:
		CountQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer,
			int count) : Query(select, join, rdfPointer) {

				this->count = count;
				this->currentCount = 0;
		}
		
		int getCurrentCount() {
			return currentCount;
		}

		void incrementCount() {
			this->currentCount++;
		}

		bool isReady() {
			if (currentCount == count) {
				currentCount = 0;
				return true;
			}
			return false;
		}
		

	
		void launch() {
			std::cout << "starting windows is " << windowPointer.rdfStore.begin << " ending window is " << windowPointer.rdfStore.end << std::endl;
			queryManager(getSelect(), getJoin(), windowPointer, count);
			windowPointer.advanceBegin(count);
			printResults();
		}
};

class TimeQuery : public Query {
	private:
		circularBuffer<long int> timestampPointer;
		int stepTime;
		int windowTime;
		long int lastTimestamp;
		bool isFirst;

	public:
		TimeQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer, circularBuffer<long int> timestampPointer,
			int windowTime, int stepTime) : Query(select, join, rdfPointer) {
				this->stepTime = stepTime;
				this->windowTime = windowTime;
				this->lastTimestamp = 0;
				this->isFirst = true;
				this->timestampPointer = timestampPointer;
		}
		
		void setStartingTimestamp(long int timestamp) {
			this->lastTimestamp = timestamp;
		}

		void advancePointer(int step)  {
			Query::advancePointer(step);
			timestampPointer.end = (timestampPointer.end + step) % timestampPointer.size;
		}

		

		bool isReady(long int newTimestamp) {
			if (isFirst) {
				
				if (lastTimestamp + windowTime <= newTimestamp) {
					return true;
				}

			} else {
				if (lastTimestamp + stepTime <= newTimestamp) {
					return true;
				}
			}
			
			return false;
		}

		long int* getTimestampPointer() {
			return this->timestampPointer.pointer;
		}

		void launch() {
			lastTimestamp += stepTime;
			if (!isFirst) {
				long int startingTime = lastTimestamp - (long int) windowTime;					
				int newBegin = 0;

				std::cout<< "comparing timestamp" << std::endl;
				
				std::cout<< "begin " << timestampPointer.begin << " end " << timestampPointer.end << std::endl;
				if (timestampPointer.end < timestampPointer.begin) {
					bool found = false;

					for(int i = timestampPointer.begin; i < timestampPointer.size; i++) {
						if (timestampPointer.pointer[i] > startingTime) {
							found = true;
							newBegin = i;
							break;
						}
					}				

					if (found == false) {
						for(int i = 0; i < timestampPointer.end; i++) {
							if (timestampPointer.pointer[i] > startingTime) {

								newBegin = i;
								break;
							}				
						}				
					}

				} else {
					for (int i = timestampPointer.begin; i < timestampPointer.end; i++) {
						if (timestampPointer.pointer[i] > startingTime) {
							newBegin = i;
							break;
						}
					}

				}

				windowPointer.setBegin(newBegin);
				timestampPointer.begin = newBegin;

				std::cout << "new begin is " << newBegin << std::endl;
				
			} else {
				isFirst = false;
			}	

			

			int diff = windowPointer.rdfStore.end - windowPointer.rdfStore.begin;
			int currentSize = (diff >= 0 ? diff : windowPointer.rdfStore.size + diff); 
			queryManager(getSelect(), getJoin(), windowPointer, currentSize);  
			printResults();
		}


};




class TripleGenerator {
	private:
		int spanTime;
		std::string fileUrl;

		std::vector<tripleContainer> rdfBuffer;
		std::vector<int> subjectBuffer;
		std::vector<int> predicateBuffer;
		std::vector<int> objectBuffer;

		std::vector<CountQuery> countQueries;
		std::vector<TimeQuery> timeQueries;

		circularBuffer<long int> timestampPointer;
		deviceCircularBuffer devicePointer;

	public:
		TripleGenerator(int spanTime, std::string fileUrl, int buffSize)   {
			this->spanTime = spanTime;
			this->fileUrl = fileUrl;
			long int* timestamp = (long int*) malloc(buffSize * sizeof(long int));
			timestampPointer.pointer = timestamp;
			timestampPointer.size = buffSize;
		}
		
		void setDevicePointer(deviceCircularBuffer devicePointer) {
			this->devicePointer = devicePointer;
		}
		
		circularBuffer<long int> getTimestampPointer() {
			return timestampPointer;
		}
		
		void addTimeQuery(TimeQuery query) {
			timeQueries.push_back(query);
		}
		
		void addCountQuery(CountQuery query) {
			countQueries.push_back(query);
		}
		
		
		void copyElements (int deviceSpan, int hostSpan, int copySize) {
			hipMemcpy(deviceSpan + devicePointer.rdfStore.pointer, &rdfBuffer[0] + hostSpan, copySize * sizeof(tripleContainer), hipMemcpyHostToDevice); 
			hipMemcpy(deviceSpan + devicePointer.subject.pointer, &subjectBuffer[0] + hostSpan, copySize * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.predicate.pointer,&predicateBuffer[0] + hostSpan, copySize * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.object.pointer, &objectBuffer[0] + hostSpan, copySize * sizeof(int), hipMemcpyHostToDevice);
		}

		
		void advanceDevicePointer() {
			int copySize = rdfBuffer.size();
			
			circularBuffer<tripleContainer> rdfBuff = devicePointer.rdfStore;

			int newEnd = (rdfBuff.end + copySize) % rdfBuff.size;

				 
			if (newEnd < rdfBuff.end) {
				int finalHalf = rdfBuff.size - rdfBuff.end;
				copyElements(devicePointer.rdfStore.end, 0, finalHalf);			
	
				int firstHalf = copySize - finalHalf;
				copyElements(0, finalHalf, firstHalf);			
			} else {
				copyElements(devicePointer.rdfStore.end, 0, copySize);	
			}
			


			devicePointer.setEnd(newEnd);

			rdfBuffer.clear();
			subjectBuffer.clear();
			predicateBuffer.clear();
			objectBuffer.clear();
		}
		
		void checkStep() {	
			for (auto &query : countQueries)  {
				query.incrementCount();
				if (query.isReady()) {
					int copySize = rdfBuffer.size();
					advanceDevicePointer();
					 
					for (auto &other : countQueries)  {
						other.advancePointer(copySize);
					}
				
					for (auto &other : timeQueries) {
						other.advancePointer(copySize);
					}
					
					
					query.launch();
				}

			}
			

			for (auto &query : timeQueries) {
				if (query.isReady(timestampPointer.pointer[timestampPointer.end - 1])) {
				
					int copySize = rdfBuffer.size();
					advanceDevicePointer();

			
					for (auto &other : countQueries)  {
						other.advancePointer(copySize);
					}
				
					for (auto &other : timeQueries) {
						if (&other == &query) {
							continue;
						}
						other.advancePointer(copySize);
					}


					std::cout << "copy size is " << copySize << std::endl;
					query.advancePointer(copySize - 1);
					query.launch();
					query.advancePointer(1);

				}
			}
		}
		
		void start() {
			std::ifstream rdfStoreFile (fileUrl);
			std::string strInput;
			
			int counter = 0;

			struct timeval startingTs;
			gettimeofday(&startingTs, NULL);
			long int ts = startingTs.tv_sec * 1000000 + startingTs.tv_usec;

			for (auto &query : timeQueries) {
				
				query.setStartingTimestamp(ts);
			}
			
			while (std::getline(rdfStoreFile, strInput)) {
			
				std::vector<std::string> triple;
		                separateWords(strInput, triple, ' ');

				tripleContainer currentTriple  = {atoi(triple[0].c_str()), atoi(triple[1].c_str()), atoi(triple[2].c_str())};
				
				
				struct timeval tp;
				gettimeofday(&tp, NULL);
				long int ms = tp.tv_sec * 1000000 + tp.tv_usec;
				
				timestampPointer.pointer[timestampPointer.end] = ms;
				timestampPointer.end = (timestampPointer.end + 1) % timestampPointer.size;
				timestampPointer.begin = timestampPointer.end;
				
				
				rdfBuffer.push_back(currentTriple);
				subjectBuffer.push_back(currentTriple.subject);
				predicateBuffer.push_back(currentTriple.predicate);
				objectBuffer.push_back(currentTriple.object);
				
				checkStep();
				
				usleep(spanTime);
			}
			
			
			//***** REMOVE IN FINAL CODE: ONLY FOR TEST *****
			int copySize = rdfBuffer.size();
			
			if (copySize != 0 ) {
				advanceDevicePointer();
				std::cout << "copy size is " << copySize << std::endl;		 
				for (auto &query : countQueries)  {
					query.advancePointer(copySize);
					query.launch();
				}
			
				for (auto &query : timeQueries) {
					query.advancePointer(copySize);
					query.launch();
				}
							
	
			}
			

			//***** END REMOVE PART-*****			
		}



};


__global__ void unarySelect (circularBuffer<int> src, int* value, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size, int storeSize) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (abs(src.end - src.begin +  src.size) % src.size) ) {
		return;
	}	

	int newIndex = (src.begin + index) % src.size;
	
	if (src.pointer[newIndex] == (*value)) {
		int add = atomicAdd(size, 1);
		dest[add] = store.pointer[newIndex];
	}
}

__global__ void binarySelect (circularBuffer<int> src1, circularBuffer<int> src2, int* value1, int* value2, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size, int storeSize) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (abs(src1.end - src1.begin + src1.size) % src1.size) ) {
		return;
	}		

	int newIndex = (src1.begin + index) % src1.size;
	if ((src1.pointer[newIndex] == (*value1)) && (src2.pointer[newIndex] == (*value2))) {
		int add = atomicAdd(size, 1);
		dest[add] = store.pointer[newIndex];
	}
}



/*
* Make multiple select query, with specified comparison condition,
* on a triple store. Both queries and the store are supposed to 
* be already on the device. 
* 
* @param d_selectQueries : the array in which are saved the select values
* @param d_storePointer : pointer on the device to the triple store
* @param storeSize : size of the triple store
* @param comparatorMask : array of triple of comparator that are applied to the queries
*			must be of the size of the d_selectQueries
* @return a vector of type mem_t in which are saved the query results.
*/
std::vector<mem_t<tripleContainer>*> rdfSelect(const std::vector<tripleContainer*> d_selectQueries, 
		deviceCircularBuffer d_pointer,
		const int storeSize, 
		std::vector<int*> comparatorMask,
		std::vector<int>  arrs) 
{
	standard_context_t context;
	//Initialize elements
	int querySize =  d_selectQueries.size();
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	int* currentSize;
	hipMalloc(&currentSize, sizeof(int));
	int* zero = (int*) malloc(sizeof(int));
	*zero = 0;

	int* finalResultSize  = (int*) malloc(sizeof(int));

	//Cycling on all the queries
	for (int i = 0; i < querySize; i++) {
		//Save variable to pass to the lambda operator
		tripleContainer* currentPointer = d_selectQueries[i];
		
		mem_t<tripleContainer>* currentResult = new mem_t<tripleContainer>(storeSize, context);

		int gridSize = 300;
	        int blockSize = (storeSize / gridSize) + 1;
		hipMemcpy(currentSize, zero, sizeof(int), hipMemcpyHostToDevice);
			
		switch(arrs[i]) {

			case(0): {
				int* value = &(currentPointer->subject);

				unarySelect<<<gridSize,blockSize>>>(d_pointer.subject, value, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}

			case(1): {
				int* value = &(currentPointer->predicate);
			
				unarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, value, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}
						
			case(2): {
                                int* value = &(currentPointer->object);

                                unarySelect<<<gridSize,blockSize>>>(d_pointer.object, value, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);
                                break;
			}
			
			case(3): {
				int* value1 = &(currentPointer->subject);
				int* value2 = &(currentPointer->predicate);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.predicate, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}

			case(4): {
				int* value1 = &(currentPointer->subject);
				int* value2 = &(currentPointer->object);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.object, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}

			case(5): {
				int* value1 = &(currentPointer->predicate);
				int* value2 = &(currentPointer->object);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, d_pointer.object, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}
						
			case(6): {
				hipMemcpy(currentResult->data(), d_pointer.rdfStore.pointer, storeSize * sizeof(tripleContainer), hipMemcpyDeviceToDevice);
				hipMemcpy(currentSize, &storeSize, sizeof(int), hipMemcpyHostToDevice);
                                break;
			}
			
			
			default: {
				printf("ERROR ERRROR ERROR ERROR ERROR ERROR ERROR");
			}


		}
				
                hipMemcpy(finalResultSize, currentSize, sizeof(int), hipMemcpyDeviceToHost);
		currentResult->setSize(*finalResultSize);
		finalResults.push_back(currentResult);
	}
	hipFree(currentSize);
	
	return finalResults;
}



//Sorter for sorting the triple due to theorder defined by the sortMask
 class TripleSorter {
	private:
		int sortMask[3];
	public:
		TripleSorter(JoinMask sortMask[3]) {
			this->sortMask[0] = static_cast<int> (sortMask[0]);
			this->sortMask[1] = static_cast<int> (sortMask[1]);
			this->sortMask[2] = static_cast<int> (sortMask[2]);
				
		}
		
		MGPU_DEVICE bool operator() (tripleContainer a, tripleContainer b) {
			int tripleA[3] = {a.subject, a.predicate, a.object};
			int tripleB[3] = {b.subject, b.predicate, b.object};
			
			if ((sortMask[0] != -1) && (tripleA[sortMask[0]] < tripleB[sortMask[0]])) {
				return true;
			}
			
			if ((sortMask[1] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] < tripleB[sortMask[1]])) {
				return true;
			}
			
			if ((sortMask[2] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] == tripleB[sortMask[1]]) && (tripleA[sortMask[2]] < tripleB[sortMask[2]])) {
				return true;
			}
			
			return false;
		}
};


//Computes inner < outer when doing join to find the elements that needs to be joined
class TripleComparator
{
	private:
		int maskA[3];
		int maskB[3];
	public:
		TripleComparator(JoinMask innerMask[3], JoinMask outerMask[3]) {
			maskA[0] = static_cast<int> (innerMask[0]);
			maskA[1] = static_cast<int> (innerMask[1]);
			maskA[2] = static_cast<int> (innerMask[2]);
			
			maskB[0] = static_cast<int> (outerMask[0]);
			maskB[1] = static_cast<int> (outerMask[1]);
			maskB[2] = static_cast<int> (outerMask[2]);			
		}
		
		MGPU_DEVICE bool operator() (tripleContainer a, tripleContainer b) {			
			int tripleA[3] = {a.subject, a.predicate, a.object};
			int tripleB[3] = {b.subject, b.predicate, b.object};

			if ((maskA[0] != -1) && (tripleA[maskA[0]] < tripleB[maskB[0]])) {
				return true;
			}
		
			if ((maskA[1] != -1) && (tripleA[maskA[0]] == tripleB[maskB[0]]) && (tripleA[maskA[1]] < tripleB[maskA[1]])) {
				return true;
			}
			
			if ((maskA[2] != -1) && (tripleA[maskA[0]] == tripleB[maskB[0]]) && (tripleA[maskA[1]] == tripleB[maskA[1]]) && (tripleA[maskA[2]] < tripleB[maskA[2]])) {
				return true;
			}
		
			return false;
		}
};


__global__ void indexCopy(tripleContainer* innerSrc, tripleContainer* innerDest, tripleContainer* outerSrc, tripleContainer* outerDest, int2* srcIndex, int maxSize) 
{
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (destIndex  >= maxSize)  {
		return;
	}
	
	//INVETERD INDEX DUE TO INVERTED JOIN PROBLEM (it should be inner = x, outer = y)
	int innerIndex = srcIndex[destIndex].y;
	int outerIndex = srcIndex[destIndex].x;
	
	innerDest[destIndex] = innerSrc[innerIndex];	
	outerDest[destIndex] = outerSrc[outerIndex];
}

std::vector<mem_t<tripleContainer>*> rdfJoin(tripleContainer* innerTable, int innerSize, tripleContainer* outerTable, int outerSize, JoinMask innerMask[3], JoinMask outerMask[3])
{
	standard_context_t context;
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	TripleSorter* innerSorter = new TripleSorter(innerMask);
	TripleSorter* outerSorter = new TripleSorter(outerMask);
	
	//Sort the two input array
	mergesort(innerTable, innerSize , *innerSorter, context);
	mergesort(outerTable, outerSize , *outerSorter, context);
	
	TripleComparator* comparator = new TripleComparator(innerMask, outerMask);
	
	//BUG che mi costringe ad invertire inner con outer?
	mem_t<int2> joinResult = inner_join(outerTable, outerSize, innerTable, innerSize,  *comparator, context);
		
	mem_t<tripleContainer>* innerResults = new mem_t<tripleContainer>(joinResult.size(), context);
        mem_t<tripleContainer>* outerResults = new mem_t<tripleContainer>(joinResult.size(), context);
	
	//SETTARE DIVISIONE CORRETTA
	//BIsogna settare come comporatrsi quando il valore della join supera i 129k risultati
	int gridSize = 64;
	int blockSize = (joinResult.size() / gridSize) + 1; 
	indexCopy<<<gridSize, blockSize>>>(innerTable, innerResults->data(), outerTable, outerResults->data(), joinResult.data(), joinResult.size());

	finalResults.push_back(innerResults);
	finalResults.push_back(outerResults);

	return finalResults;
}



/**
* Function for managing query execution
**/
void queryManager(std::vector<SelectOperation*> selectOp, std::vector<JoinOperation*> joinOp, 
		deviceCircularBuffer d_pointer, 
		const int storeSize) {

	std::vector<tripleContainer*> d_selectQueries;
	std::vector<int*> comparatorMask;
	std::vector<int> arrs;

	for (int i = 0; i < selectOp.size(); i++) {
		d_selectQueries.push_back(selectOp[i]->getQuery()->data());
		arrs.push_back(selectOp[i]->getArr());
	}
	

	std::vector<mem_t<tripleContainer>*> selectResults = rdfSelect(d_selectQueries, d_pointer, storeSize, comparatorMask, arrs);

	for (int i = 0; i < selectResults.size(); i++) {
		selectOp[i]->setResult(selectResults[i]);
	}
	
	
	for (int i = 0; i < joinOp.size(); i++) {
		mem_t<tripleContainer>* innerTable = *joinOp[i]->getInnerTable();
		mem_t<tripleContainer>* outerTable = *joinOp[i]->getOuterTable();
		std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin(innerTable->data(), innerTable->size(), outerTable->data(), outerTable->size(), joinOp[i]->getInnerMask(), joinOp[i]->getOuterMask());
		joinOp[i]->setInnerResult(joinResult[0]);
		joinOp[i]->setOuterResult(joinResult[1]);				
	}
	
}




template<typename type_t, typename accuracy>
std::vector<accuracy> stats(std::vector<type_t> input) {
	int size = input.size();
	float mean = 0;
	float variance = 0;
	for (int i = 0; i < size; i++) {
		mean += (accuracy) input[i];
                variance += (accuracy)  (input[i] * input[i]);
        }
        mean = mean / ((accuracy) size);
        variance = variance / ((accuracy) size);
        variance = variance - (mean * mean);
        std::vector<accuracy> statistic;
	statistic.push_back(mean);
	statistic.push_back(variance);
	return statistic;
}

int main(int argc, char** argv) {
 
		using namespace std;
		struct timeval beginPr, beginCu, beginEx, end;
		gettimeofday(&beginPr, NULL);	
		hipDeviceReset();
		standard_context_t context;

                size_t BUFFER_SIZE = 400000;
		deviceCircularBuffer windowPointer;


		std::vector<float> timeCuVector;                
		std::vector<float> timeExVector;

		TripleGenerator manager(1, argv[1], BUFFER_SIZE);

                int N_CYCLE = 1;
		for (int i = 0; i < N_CYCLE; i++) {
			gettimeofday(&beginCu, NULL);
			
			hipMalloc(&windowPointer.rdfStore.pointer, BUFFER_SIZE * sizeof(tripleContainer));
			hipMalloc(&windowPointer.subject.pointer, BUFFER_SIZE * sizeof(int));
                        hipMalloc(&windowPointer.predicate.pointer, BUFFER_SIZE * sizeof(int));
                        hipMalloc(&windowPointer.object.pointer, BUFFER_SIZE * sizeof(int));
					
			int begin = 0;
			windowPointer.setBegin(begin);
			windowPointer.setEnd(begin);
			windowPointer.setSize(BUFFER_SIZE);
			
			manager.setDevicePointer(windowPointer);
			
		        //set Queries (select that will be joined)
		        tripleContainer h_queryVector1 =  {-1, -1 , 99};
		        tripleContainer h_queryVector2 =	{0, -1, -1}; 

			cout << "query is: first obj: " << h_queryVector1.object << "; secons subj: " << h_queryVector2.subject << endl;

		        mem_t<tripleContainer> d_queryVector1(1, context);
			hipMemcpy(d_queryVector1.data(), &h_queryVector1, sizeof(tripleContainer), hipMemcpyHostToDevice);
		
		        mem_t<tripleContainer> d_queryVector2(1, context);
			hipMemcpy(d_queryVector2.data(), &h_queryVector2, sizeof(tripleContainer), hipMemcpyHostToDevice);
	
			//set select mask operation
			std::vector<tripleContainer*> selectQuery;
			selectQuery.push_back(d_queryVector1.data());
			selectQuery.push_back(d_queryVector2.data());

			SelectArr arr1 = SelectArr::O;
			SelectArr arr2 = SelectArr::S;
		
			//set Join mask
			JoinMask innerMask[3];
			innerMask[0] = JoinMask::PRE;
			innerMask[1] = JoinMask::NJ;
			innerMask[2] = JoinMask::NJ;
			
			JoinMask outerMask[3];
			outerMask[0] = JoinMask::PRE;
			outerMask[1] = JoinMask::NJ;
			outerMask[2] = JoinMask::NJ;

			//Creat operation object to pass to query manager
			SelectOperation  selectOp1(&d_queryVector1, arr1);
			SelectOperation  selectOp2(&d_queryVector2, arr2);
		
			JoinOperation  joinOp(selectOp1.getResultAddress(), selectOp2.getResultAddress(), innerMask, outerMask);
		
			std::vector<SelectOperation*> selectOperations;
			std::vector<JoinOperation*> joinOperations;
		
			selectOperations.push_back(&selectOp1);
			selectOperations.push_back(&selectOp2);
			joinOperations.push_back(&joinOp);
			
			int stepCount = 100000;

			/*TimeQuery count(selectOperations, joinOperations, windowPointer, manager.getTimestampPointer(), 6000000, 6000000);
			manager.addTimeQuery(count);*/

			CountQuery count(selectOperations, joinOperations, windowPointer, 150000);
			manager.addCountQuery(count);
			
			gettimeofday(&beginEx, NULL);	
			
			manager.start();

			
			/*//Retrive results from memory
			std::vector<tripleContainer> finalInnerResults = from_mem(*joinOp.getInnerResult());
			std::vector<tripleContainer> finalOuterResults = from_mem(*joinOp.getOuterResult());*/
			
			hipDeviceSynchronize();
			gettimeofday(&end, NULL);

                      /*  std::vector<tripleContainer> selectResults = from_mem(*selectOp1.getResult());
                        std::vector<tripleContainer> selectResults2 = from_mem(*selectOp2.getResult());*/

			float exTime = (end.tv_sec - beginEx.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginEx.tv_usec) / 1000 ;
			float prTime = (end.tv_sec - beginPr.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginPr.tv_usec) / 1000 ;
			float cuTime = (end.tv_sec - beginCu.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginCu.tv_usec) / 1000 ;
			
			timeCuVector.push_back(cuTime);
			timeExVector.push_back(exTime);
			/*
			//Print Results
			cout << "first select result" << endl;
			for (int i = 0; i < selectResults.size(); i++) {
				cout << selectResults[i].subject << " " << selectResults[i].predicate << " "  << selectResults[i].object << endl; 
			}*/
			

			
			//Print current cycle results
			/*cout <<"first Select Size " << selectResults.size() << endl;
			cout << "second Select Size " << selectResults2.size() << endl;
			cout << "join Size " << finalOuterResults.size() << endl;*/
						
			cout << "Total time: " << prTime << endl;
			cout << "Cuda time: " << cuTime << endl;
			cout << "Execution time: " << exTime << endl;					
			cout << "" << endl;


			hipFree(windowPointer.subject.pointer);
			hipFree(windowPointer.predicate.pointer);
			hipFree(windowPointer.object.pointer);
			hipFree(windowPointer.rdfStore.pointer);
		}

/*	
		std::vector<float> statistics;
		
		statistics = stats<float, float>(timeCuVector);	
                cout << "mean cuda time " << statistics[0] << endl;
                cout << "variance cuda time " << statistics[1] << endl;

                statistics = stats<float, float>(timeExVector);
                cout << "mean ex time " << statistics[0] << endl;
                cout << "variance ex time " << statistics[1] << endl;
*/
                return 0;
}
