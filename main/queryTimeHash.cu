#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <unistd.h>
#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_join.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <sys/time.h>
#include <sparsehash/dense_hash_map>

using namespace mgpu;
using google::dense_hash_map;


//TODO implementare la projection su gpu

//TODO 
//VARIABILI PER TESTING, DA RIMUOVERE DAL CODICE FINALE
int VALUE = 0;
std::vector<float> timeCuVector;                
std::vector<long int> timeExVector;
bool isLaunched = false;
dense_hash_map<size_t, std::string> map;

//** END TESTING ***//

//struct to contains a single triple with int type.
struct tripleContainer {
        size_t subject;
        size_t predicate;
        size_t object;

	void print() {
		std::cout << subject << " " << predicate << " " << object << std::endl;
	}
};

//Struct for circular buffer
template<typename type_t>
struct circularBuffer {
	type_t* pointer;
	int begin;
	int end;
	int size;
	
	circularBuffer() : pointer(0), begin(0), end(0), size(0) {}
	
	int getLength() {
		return (abs(end - begin + size) % size);
	}
};

//Struct for containing the pointer to an rdf store (divided into subject predicate and object) 
template<typename rdf_t, typename arr_t>
struct triplePointer {
	rdf_t rdfStore;
	arr_t subject;
	arr_t predicate;
	arr_t object;
};


template <int N>
struct Row 
{
        size_t element[N];
};


struct Binding {
	size_t* pointer;
	int width;
	int height;
	std::vector<std::string> header;
	
	Binding() {
	
	}
	
	Binding(int width, int height) {
		hipMalloc(&pointer, width * height *  sizeof(size_t));
		this->width = width;
		this->height = height;
	}
};


/*
* Specific implementation of triplePointer for ciruclar buffer.
* Offers methods for managing the attributes of the class.
*/
struct deviceCircularBuffer : triplePointer<circularBuffer<tripleContainer>, circularBuffer<size_t>> {
	void setValues(int begin, int end, int size) {
		setBegin(begin);
		setEnd(end);
		setSize(size);
	}	

	void setBegin(int begin) {
		rdfStore.begin = begin;
		subject.begin = begin;
		predicate.begin = begin;
		object.begin = begin;
	}
	
	void setEnd(int end) {
		rdfStore.end = end;
		subject.end = end;
		predicate.end = end;
		object.end = end;
	}
	
	void setSize(int size) {
		rdfStore.size = size;
		subject.size = size;
		predicate.size = size;
		object.size = size;
	}
	
	void advanceBegin(int step){
		int newBegin = (rdfStore.begin + step) % rdfStore.size;
		setBegin(newBegin);
	}
				
};

int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}

/*
* Join enum to define order and which element to join
* NJ indicates a non-join value, so it is ignored during join and sorting
* So that it improves performance avoiding uneecessary conditional expression
*/
enum class JoinMask {NJ = -1, SBJ = 0, PRE = 1, OBJ = 2};


//Section for defining operation classes
class JoinOperation 
{	
	//TODO Modificare classe in modo che permetta la join di join
	private:
		Binding** innerTable;
		Binding** outerTable;
		Binding* result = 0;
		
		std::vector<std::string> joinMask;

	public:
		JoinOperation(Binding** innerTable, Binding** outerTable, std::string joinMask) {
			this->innerTable = innerTable;
			this->outerTable = outerTable;
			separateWords(joinMask, this->joinMask, ' ');
		};
			
		Binding* getInnerTable() {
			return *this->innerTable;
		};
		
		Binding* getOuterTable() {
			return *this->outerTable;
		};
		
		std::vector<std::string> getJoinMask() {
			return this->joinMask;
		};
		

		Binding* getResult() {
			return this->result;
		};
		
		void setResult(Binding* result) {
			this->result = result;
		};
		

		
		
};

enum class SelectArr { S = 0, P = 1, O = 2, SP = 3, SO = 4, PO = 5, SPO = 6};




__global__ void unarySelect (circularBuffer<tripleContainer> src, int target, int first, int second, size_t* value, size_t* dest, int width, int* size) {

			int index = blockIdx.x * blockDim.x + threadIdx.x;

			if (index >= (abs(src.end - src.begin +  src.size) % src.size) ) {
				return;
			}	

			int newIndex = (src.begin + index) % src.size;
	
			size_t temp[3] = {src.pointer[newIndex].subject, src.pointer[newIndex].predicate, src.pointer[newIndex].object};
	
			if (temp[target] == (*value)) {
				int add = atomicAdd(size, 1);
				size_t* dest_p = (size_t*) (dest + add * width) ;



				*dest_p = temp[first];
				*(dest_p + 1) = temp[second];
		
			}
		}


		__global__ void binarySelect (circularBuffer<size_t> src1, circularBuffer<size_t> src2, size_t* value1, size_t* value2, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size) {

			int index = blockIdx.x * blockDim.x + threadIdx.x;
		
			if (index >= (abs(src1.end - src1.begin + src1.size) % src1.size) ) {
				return;
			}		

			int newIndex = (src1.begin + index) % src1.size;
			if ((src1.pointer[newIndex] == (*value1)) && (src2.pointer[newIndex] == (*value2))) {
				int add = atomicAdd(size, 1);
				dest[add] = store.pointer[newIndex];
		
			}
		}



class SelectOperationInteface {
	protected:
		mem_t<tripleContainer>* query;
		Binding* result;
		int arr;
		std::vector<std::string> variables;

	public:
		SelectOperationInteface (mem_t<tripleContainer>* query, SelectArr arr, std::string variable) {
			this->query = query;	
			this->arr = static_cast<int> (arr);
			separateWords(variable, variables, ' ');
		};

		int getArr() {
			return this-> arr;
		}
			
		mem_t<tripleContainer>* getQuery() {
			return this->query;
		};
		                                                                            
		
		void setResult(Binding* result) {
			this->result = result;
		};
		
		Binding* getResult() {
			return result;
		};
		
		std::vector<std::string> getVariables() {
			return variables;
		};
		
		Binding** getResultAddress() {
			return &result;
		}


		





		/*
		* Make multiple select query, with specified comparison condition,
		* on a triple store. Both queries and the store are supposed to 
		* be already on the device. 
		* 
		* @param d_selectQueries : the array in which are saved the select values
		* @param d_storePointer : pointer on the device to the triple store
		* @param storeSize : size of the triple store
		* @return a vector of type mem_t in which are saved the query results.
		*/
		void rdfSelect(deviceCircularBuffer d_pointer, const int storeSize) 
		{	
			
		}


};



class SelectOperation 
{
	private:
		mem_t<tripleContainer>* query;
		Binding* result;
		int arr;
		std::vector<std::string> variables;

	public:
		SelectOperation(mem_t<tripleContainer>* query, SelectArr arr, std::string variable) {
			this->query = query;	
			this->arr = static_cast<int> (arr);
			separateWords(variable, variables, ' ');
		};

		int getArr() {
			return this-> arr;
		}
			
		mem_t<tripleContainer>* getQuery() {
			return this->query;
		};
		                                                                            
		
		void setResult(Binding* result) {
			this->result = result;
		};
		
		Binding* getResult() {
			return result;
		};
		
		std::vector<std::string> getVariables() {
			return variables;
		};
		
		Binding** getResultAddress() {
			return &result;
		}


		





		/*
		* Make multiple select query, with specified comparison condition,
		* on a triple store. Both queries and the store are supposed to 
		* be already on the device. 
		* 
		* @param d_selectQueries : the array in which are saved the select values
		* @param d_storePointer : pointer on the device to the triple store
		* @param storeSize : size of the triple store
		* @return a vector of type mem_t in which are saved the query results.
		*/
		void rdfSelect(deviceCircularBuffer d_pointer, const int storeSize) 
		{	
			//Initialize elements	
			int* d_resultSize;
			hipMalloc(&d_resultSize, sizeof(int));
			int h_resultSize  = 0;

			hipMemcpy(d_resultSize, &h_resultSize, sizeof(int), hipMemcpyHostToDevice);
	
			//INSERIRE DIVISIONE CORRETTA
			int gridSize = 300;
			int blockSize = (storeSize / gridSize) + 1;
			tripleContainer* query = this->query->data();
		
			result = new Binding(2, storeSize);
			std::cout << "TARGET ADDRESS IS " << result << std::endl;	
			switch(arr) {

				case(0): {
					size_t* value = &(query->subject);
					unarySelect<<<gridSize,blockSize>>>(d_pointer.rdfStore, 0, 1, 2, value, result->pointer, result->width, d_resultSize);
					break;
				}

				case(1): {
					size_t* value = &(query->predicate);
					unarySelect<<<gridSize,blockSize>>>(d_pointer.rdfStore,  1, 0, 2, value, result->pointer, result->width, d_resultSize);
					break;
				}
					
				case(2): {
				
			                size_t* value = &(query->object);
	
			                unarySelect<<<gridSize,blockSize>>>(d_pointer.rdfStore,  2, 0, 1, value, result->pointer, result->width, d_resultSize);
			             
			                break;
				}
		
		/*		case(3): {
					size_t* value1 = &(query->subject);
					size_t* value2 = &(query->predicate);
					binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.predicate, value1, value2, result->data(), d_pointer.rdfStore, d_resultSize);
					break;
				}

				case(4): {
					size_t* value1 = &(query->subject);
					size_t* value2 = &(query->object);
					binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.object, value1, value2, result->data(), d_pointer.rdfStore, d_resultSize);
					break;
				}

				case(5): {
					size_t* value1 = &(query->predicate);
					size_t* value2 = &(query->object);
					binarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, d_pointer.object, value1, value2, result->data(), d_pointer.rdfStore, d_resultSize);
					break;
				}
					
				case(6): {
					hipMemcpy(result->data(), d_pointer.rdfStore.pointer, storeSize * sizeof(tripleContainer), hipMemcpyDeviceToDevice);
					hipMemcpy(d_resultSize, &storeSize, sizeof(int), hipMemcpyHostToDevice);
			                break;
				}*/
		
			}
			hipDeviceSynchronize();
	
	
			hipMemcpy(&h_resultSize, d_resultSize, sizeof(int), hipMemcpyDeviceToHost);

			result->height  =  h_resultSize;		

			hipFree(d_resultSize);

		}




	
};

//Sorter for sorting the triple due to theorder defined by the sortMask
template<int N>
 class Sorter {
	private:
		int sortMask[3];
	public:
		Sorter(int sortMask[3]) {
			this->sortMask[0] =  (sortMask[0]);
			this->sortMask[1] =  (sortMask[1]);
			this->sortMask[2] =  (sortMask[2]);
				
		}
		
		
		MGPU_DEVICE bool operator() (Row<N> a, Row<N> b) {
			        	
			if ((sortMask[0] != -1) && (a.element[sortMask[0]] < b.element[sortMask[0]])) {
				return true;
			}
			
			if ((sortMask[1] != -1) && (a.element[sortMask[0]] == b.element[sortMask[0]]) && (a.element[sortMask[1]] < b.element[sortMask[1]])) {
				return true;
			}
			
			if ((sortMask[2] != -1) && (a.element[sortMask[0]] == b.element[sortMask[0]]) && (a.element[sortMask[1]] == b.element[sortMask[1]]) && (a.element[sortMask[2]] < b.element[sortMask[2]])) {
				return true;
			}
			
			return false;
		}
};


 
template<int N>
__global__ void typeCopy(Row<N>* dest, size_t* src, int* pos, int maxSize, int width) {
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (destIndex  >= maxSize)  {
		return;
	}

	dest[threadIdx.x].element[pos[blockIdx.x]] = src[threadIdx.x * width + blockIdx.x];
}



__global__ void indexCopy(/*tripleContainer* innerSrc, tripleContainer* innerDest, tripleContainer* outerSrc, tripleContainer* outerDest, int2* srcIndex, int maxSize*/) 
{
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	printf("BOH \n");
 /*
	if (destIndex  >= maxSize)  {
		return;
	}
	/*
	int innerIndex = srcIndex[destIndex].x;
	int outerIndex = srcIndex[destIndex].y;
	
	innerDest[destIndex] = innerSrc[innerIndex];	
	outerDest[destIndex] = outerSrc[outerIndex];*/
}

template<int N>
__global__ void printTest(Row<N>* src1, Row<N>* src2, int2* indexes) {

	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	printf("INDeXES ARE %i %i VALUES ARE %i  %i %i %i \n", indexes[destIndex].x, indexes[destIndex].y, src1[indexes[destIndex].x].element[0], src1[indexes[destIndex].x].element[1], src2[indexes[destIndex].y].element[0], src2[indexes[destIndex].y].element[1]);
}



template<int N>
std::vector<mem_t<tripleContainer>*> rdfJoin(Binding* innerTable, Binding* outerTable, std::vector<std::string> joinMask)
{
	//TODO Migliorare la join nel reordering delle triple
	standard_context_t context;
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	Row<N>* tempInner;
	hipMalloc(&tempInner, innerTable->height * sizeof(Row<N>));
	
	Row<N>* tempOuter;
	hipMalloc(&tempOuter, outerTable->height * sizeof(Row<N>));
	
	int* d_innerPos;
	hipMalloc(&d_innerPos, innerTable->width * sizeof(int));
	int* d_outerPos;
	hipMalloc(&d_outerPos, outerTable->width * sizeof(int));
	
	int* h_innerPos = (int*) malloc(innerTable->width * sizeof(int));
	h_innerPos[0] = 0;
	h_innerPos[1] = 1;
	int* h_outerPos = (int*) malloc(outerTable->width * sizeof(int));
	h_outerPos[0] = 1;
	h_outerPos[1] = 0;

	hipMemcpy(d_innerPos, h_innerPos, innerTable->width * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_outerPos, h_outerPos, outerTable->width * sizeof(int), hipMemcpyHostToDevice);
	
	
	typeCopy<<<innerTable->width, innerTable->height>>>(tempInner, innerTable->pointer, d_innerPos, innerTable->height * innerTable->width, innerTable->width);
	typeCopy<<<outerTable->width, outerTable->height>>>(tempOuter, outerTable->pointer, d_outerPos, outerTable->height * outerTable->width, outerTable->width);
	
		
	std::cout << "LAUNCH MEGER " << std::endl;	
	int mask[3] = {0, -1, -1};
	
	Sorter<N>* sorter = new Sorter<N>(mask);
		

	
	std::cout << "LAUNCH MEGER " << std::endl;

	//Sort the two input array
	mergesort<launch_params_t<128, 2>>(tempInner, innerTable->height, *sorter, context);
	mergesort<launch_params_t<128, 2>>(tempOuter, outerTable->height , *sorter, context);

	std::cout << "OUTER HEIGH IS " << innerTable->height << std::endl;
	Binding* d_result = innerTable;
	size_t* final_binding = (size_t*) malloc(d_result->height * d_result->width * sizeof(size_t));
	hipMemcpy(final_binding, d_result->pointer, d_result->width * sizeof(size_t) * d_result->height, hipMemcpyDeviceToHost);
	
	for (int i =0; i <10; i+=2) {
		std::cout << "original is " << map[ final_binding[i]] << " " <<  map[final_binding[i + 1]] << std::endl;
	}	
	
	Row<N>* testR  = (Row<N>*) malloc(innerTable->height * sizeof(Row<N>)); 
	hipMemcpy(testR, tempInner, innerTable->height * sizeof(Row<N>), hipMemcpyDeviceToHost);
	for (int i = 0; i < 5; i++) {
//		std::cout << "result is " << map[testR[i].element[0]] << " second " <<  map[testR[i].element[1]] << std::endl;
		std::cout << "result is " << testR[i].element[0] << " second " <<  testR[i].element[1] << std::endl;
	}
	


	mem_t<int2> joinResult = inner_join<launch_params_t<128,2>>( tempInner, innerTable->height, tempOuter, outerTable->height,  *sorter, context);
		
	std::cout << "JOIN RESULT SIZE IS " << joinResult.size() << std::endl;
	printTest<<<1, joinResult.size()>>>(tempInner, tempOuter, joinResult.data());

	
	hipDeviceSynchronize();
	std::cout << "LAUNCHED"<< std::endl;
	exit(1);
	/*
	mem_t<tripleContainer>* innerResults = new mem_t<tripleContainer>(joinResult.size(), context);
    mem_t<tripleContainer>* outerResults = new mem_t<tripleContainer>(joinResult.size(), context);
	
	//SETTARE DIVISIONE CORRETTA
	//TODO BIsogna settare come comporatrsi quando il valore della join supera i 129k risultati
	gridSize = 64;
	blockSize = (joinResult.size() / gridSize) + 1; 
	indexCopy<<<gridSize, blockSize>>>(innerTable, innerResults->data(), tempOuter->data(), outerResults->data(), joinResult.data(), joinResult.size());
	
	hipFree(tempOuter);
	hipFree(tempInner);
	
	*/

	return finalResults;
}


class Query {
	protected:
		std::vector<SelectOperation*> select;
		std::vector<JoinOperation*> join;
		deviceCircularBuffer windowPointer;

	public:
		Query(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join, deviceCircularBuffer rdfPointer) {
			this->join = join;
			this->select = select;
			this->windowPointer = rdfPointer;
		}

		virtual void setWindowEnd(int step) {
			windowPointer.setEnd(step);
		}
		
		/**
		* Function for managing query execution
		**/
		//TODO Verificare se si puo migliorare
		void startQuery() {
			int storeSize =  windowPointer.rdfStore.getLength();			
			
			for (auto op : select) {
					op->rdfSelect(windowPointer, storeSize);

				}
			
	
			for (int i = 0; i < join.size(); i++) {
				Binding* innerTable = join[i]->getInnerTable();
				Binding* outerTable = join[i]->getOuterTable();
			
				int width = 2;
			
				switch(width) {
					case(1): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<1>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(2): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<2>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}
/*
					case(3): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<3>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(4): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<4>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(5): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<5>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(6): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<6>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(7): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<7>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(8): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<8>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(9): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<9>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}

					case(10): {
						std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin<10>(innerTable, outerTable, join[i]->getJoinMask());
						break;
					}*/
				}

			/*	join[i]->setInnerResult(joinResult[0]);
				join[i]->setOuterResult(joinResult[1]);	*/		
			}
			
			exit(1);
	
		}

		//TODO modificare quando si sapra come utilizzare i risultati
		void printResults(dense_hash_map<size_t, std::string> map) {

			int w = 0;
			for (auto op : select) {
				printf("W VALUE IS %i \n", w);
				if (w == 0) VALUE += op->getResult()->height;
				
				Binding* d_result = op->getResult();
				
				size_t* final_binding = (size_t*) malloc(d_result->height * d_result->width * sizeof(size_t));
				hipMemcpy(final_binding, d_result->pointer, d_result->width * sizeof(size_t) * d_result->height, hipMemcpyDeviceToHost);
				std::cout << "size is " << d_result->height << std::endl;
				std::cout << "width is " << d_result->width << std::endl;
				
				/*for (int z = 0; z < d_result->header.size(); z++) {
				std::cout << "header are " << d_result->header[z] << std::endl;
				}*/
				
				for (int i =0; i < d_result->height; i++) {
					//for (int k = 0; k < d_result->width; k++) {
						//size_t current = final_binding[i + k];
						std::cout << "result is " << map[ final_binding[i]] << " " <<  map[final_binding[i + 1]] << std::endl;
					//}
					
				}	
			
				w++;
				hipFree(d_result->pointer);
				
				

			}
	

		/*	for (auto op : join) {
				std::cout << "join result is " << op->getInnerResult()->size() << std::endl;

				std::vector<tripleContainer> innerRes = from_mem(*op->getInnerResult());
				std::vector<const char*> innerHash;

				for (int i =0; i < innerRes.size(); i++) {
					innerHash.push_back( map[innerRes[i].subject]);
                                        innerHash.push_back( map[innerRes[i].predicate]);
                                        innerHash.push_back( map[innerRes[i].object]);

				}
				
				std::vector<tripleContainer> outerRes = from_mem(*op->getOuterResult());
				std::vector<const char*> outerHash;
				for (int i =0; i< outerRes.size(); i++) {
					outerHash.push_back( map[outerRes[i].subject]);
                                        outerHash.push_back( map[outerRes[i].predicate]);
                                        outerHash.push_back( map[outerRes[i].object]);

				}

				VALUE += op->getInnerResult()->size();
				hipFree(op->getInnerResult()->data());
				hipFree(op->getOuterResult()->data());
			}*/
					
		}
		
	
};


class CountQuery : public Query {
	private:
		int count;
		int currentCount;

	public:
		CountQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer,
			int count) : Query(select, join, rdfPointer) {

				this->count = count;
				this->currentCount = 0;
		}
		
		void incrementCount() {
			this->currentCount++;
		}
		
		bool isReady() {
			return (currentCount == count);
		}
		
		void launch() {
			startQuery();
			windowPointer.advanceBegin(count);
			currentCount = 0;
		}
		
		~CountQuery() {}
};

class TimeQuery : public Query {
	private:
		circularBuffer<long int> timestampPointer;
		long int stepTime;
		long int windowTime;
		long int lastTimestamp;
		
	public:
		TimeQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			deviceCircularBuffer rdfPointer, circularBuffer<long int> timestampPointer,
			int windowTime, int stepTime) : Query(select, join, rdfPointer) {
				this->stepTime = stepTime;
				this->windowTime = windowTime;
				this->lastTimestamp = 0;
				this->timestampPointer = timestampPointer;
		}
		
		void setWindowEnd(int step)  {
			Query::setWindowEnd(step);
			timestampPointer.end = step;
		}

		bool isReady(long int newTimestamp) {
			return (lastTimestamp + windowTime < newTimestamp);
		}

		void setStartingTimestamp(long int timestamp) {
			this->lastTimestamp = timestamp;
		}

		void launch() {	
			//Update new starting value of buffer
			int newBegin = 0;
			for(int i = timestampPointer.begin; i  != timestampPointer.end; i = (i + 1) % timestampPointer.size) {	
				if (timestampPointer.pointer[i] > lastTimestamp) {
					newBegin = i;
					break;
				}				
			}				
			windowPointer.setBegin(newBegin);
			timestampPointer.begin = newBegin;
			
			//Lancuh query and print results
			startQuery();
	
			//Update window timestamp value
			lastTimestamp += stepTime;
		}

		~TimeQuery() {}
};



template <std::size_t FnvPrime, std::size_t OffsetBasis>
struct basic_fnv_1
{
    std::size_t operator()(std::string const& text) const
    {
        std::size_t hash = OffsetBasis;
         for(std::string::const_iterator it = text.begin(), end = text.end();
                 it != end; ++it)
         {
             hash *= FnvPrime;
             hash ^= *it;
         }
         return hash;

     }
};



class QueryManager {
	private:
		int spanTime;
		std::string* source;
		int srcSize;
		
		std::vector<tripleContainer> rdfBuffer;
		std::vector<size_t> subjectBuffer;
		std::vector<size_t> predicateBuffer;
		std::vector<size_t> objectBuffer;

		std::vector<TimeQuery> timeQueries;
		std::vector<CountQuery> countQueries;
		
		circularBuffer<long int> timestampPointer;
		deviceCircularBuffer devicePointer;
    //            dense_hash_map<size_t, std::string> map;


	public:
		QueryManager(std::string* source, int srcSize, int spanTime,int buffSize)   {
			this->spanTime = spanTime;
			this->srcSize = srcSize;
			this->source = source;
			
			long int* timestamp = (long int*) malloc(buffSize * sizeof(long int));
			timestampPointer.pointer = timestamp;
			timestampPointer.size = buffSize;
		}
		
		void setDevicePointer(deviceCircularBuffer devicePointer) {
			this->devicePointer = devicePointer;
		}
		
		circularBuffer<long int> getTimestampPointer() {
			return timestampPointer;
		}
				
		void addTimeQuery(TimeQuery query) {
			timeQueries.push_back(query);
		}
		
		void addCountQuery(CountQuery query) {
			countQueries.push_back(query);
		}

				
		void copyElements (int deviceSpan, int hostSpan, int copySize) {
			hipMemcpy(deviceSpan + devicePointer.rdfStore.pointer, &rdfBuffer[0] + hostSpan, copySize * sizeof(tripleContainer), hipMemcpyHostToDevice); 
			hipMemcpy(deviceSpan + devicePointer.subject.pointer, &subjectBuffer[0] + hostSpan, copySize * sizeof(size_t), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.predicate.pointer,&predicateBuffer[0] + hostSpan, copySize * sizeof(size_t), hipMemcpyHostToDevice);
			hipMemcpy(deviceSpan + devicePointer.object.pointer, &objectBuffer[0] + hostSpan, copySize * sizeof(size_t), hipMemcpyHostToDevice);
		}

		
		void advanceDevicePointer() {
			int copySize = rdfBuffer.size();
			
			circularBuffer<tripleContainer> rdfBuff = devicePointer.rdfStore;

			int newEnd = (rdfBuff.end + copySize) % rdfBuff.size;
	 
			if (newEnd < rdfBuff.end) {
				int finalHalf = rdfBuff.size - rdfBuff.end;
				copyElements(devicePointer.rdfStore.end, 0, finalHalf);			
	
				int firstHalf = copySize - finalHalf;
				copyElements(0, finalHalf, firstHalf);			
			} else {
				copyElements(devicePointer.rdfStore.end, 0, copySize);	
			}

			devicePointer.setEnd(newEnd);

			rdfBuffer.clear();
			subjectBuffer.clear();
			predicateBuffer.clear();
			objectBuffer.clear();
		}
		
		void checkStep() {	
			for (auto &query : countQueries)  {
				query.incrementCount();
				if (query.isReady()) {
					advanceDevicePointer();
					query.setWindowEnd(devicePointer.rdfStore.end);			
					query.launch();
					query.printResults(map);
				}
			}
			
			for (auto &query : timeQueries) {
				if (query.isReady(timestampPointer.pointer[timestampPointer.end - 1])) {
					advanceDevicePointer();
					query.setWindowEnd(devicePointer.rdfStore.end - 1);		
					query.launch();
					query.printResults(map);
					query.setWindowEnd(1);
				}				
			}
		}
		
		void start() {
			struct timeval startingTs;
			gettimeofday(&startingTs, NULL);
			long int ts = startingTs.tv_sec * 1000000 + startingTs.tv_usec;

			for (auto &query : timeQueries) {
				query.setStartingTimestamp(ts);
			}
			
			usleep(1);

			basic_fnv_1< 1099511628211u, 14695981039346656037u> h_func;

			map.set_empty_key(NULL);       
		

			for (int i =0; i <srcSize; i++) {

				
				tripleContainer currentTriple;
 
                                std::vector<std::string> triple;
                                separateWords(source[i], triple, ' ');
			
			        currentTriple.subject = h_func(triple[0]);
                                currentTriple.predicate = h_func(triple[1]);
                                currentTriple.object = h_func(triple[2]);

				map[currentTriple.subject] = triple[0];
                                map[currentTriple.predicate] = triple[1];
                                map[currentTriple.object] = triple[2] ;

				struct timeval tp;
				gettimeofday(&tp, NULL);
				long int ms = tp.tv_sec * 1000000 + tp.tv_usec;


				timestampPointer.pointer[timestampPointer.end] = ms;
				timestampPointer.end = (timestampPointer.end + 1) % timestampPointer.size;
				timestampPointer.begin = timestampPointer.end;
				
				rdfBuffer.push_back(currentTriple);
				subjectBuffer.push_back(currentTriple.subject);
				predicateBuffer.push_back(currentTriple.predicate);
				objectBuffer.push_back(currentTriple.object);
							
				checkStep();
			//	usleep(spanTime);

			}
			
			//TODO vedere se occorre tenere o no quest'ultima parte
			//***** REMOVE IN FINAL CODE: ONLY FOR TEST (FORSE) *****
			//DA OTTIMIZZARE POICHE VIENE LANCIATO ANCHE QUANDO NON SERVE!!
			advanceDevicePointer();	 
			for (auto &query : timeQueries)  {
				query.setWindowEnd(devicePointer.rdfStore.end);
				query.launch();
				query.printResults(map);
			}
			
			for (auto &query :countQueries) {
				query.setWindowEnd(devicePointer.rdfStore.end);
				query.launch();
				query.printResults(map);
			}
			//***** END REMOVE PART-*****			
		}



};





template<typename type_t, typename accuracy>
std::vector<accuracy> stats(std::vector<type_t> input) {
	int size = input.size();
	float mean = 0;
	float variance = 0;
	for (int i = 0; i < size; i++) {
		mean += (accuracy) input[i];
                variance += (accuracy)  (input[i] * input[i]);
        }
        mean = mean / ((accuracy) size);
        variance = variance / ((accuracy) size);
        variance = variance - (mean * mean);
        std::vector<accuracy> statistic;
	statistic.push_back(mean);
	statistic.push_back(variance);
	return statistic;
}

int main(int argc, char** argv) {
 
		using namespace std;
		
		struct timeval beginPr, beginCu, beginEx, end;
		gettimeofday(&beginPr, NULL);	
		hipDeviceReset();
		standard_context_t context;

                size_t BUFFER_SIZE = 400000;
		deviceCircularBuffer windowPointer;




		ifstream rdfStoreFile (argv[1]);
		string strInput;

		int fileLength = 0;	 
		while (std::getline(rdfStoreFile, strInput)) {
			++fileLength;
		}
	
		rdfStoreFile.clear();
		rdfStoreFile.seekg(0, ios::beg);

                size_t rdfSize = fileLength  * sizeof(std::string);
                std::string* h_rdfStore = (std::string*) malloc(rdfSize);


                //read store from rdfStore
                for (int i = 0; i <fileLength; i++) {
			getline(rdfStoreFile,strInput);
                        h_rdfStore[i]  = strInput;
                }

                rdfStoreFile.close();

                int N_CYCLE = 1;
		for (int i = 0; i < N_CYCLE; i++) {
			
			gettimeofday(&beginCu, NULL);
			QueryManager manager(h_rdfStore, fileLength, 1, BUFFER_SIZE);
			hipMalloc(&windowPointer.rdfStore.pointer, BUFFER_SIZE * sizeof(tripleContainer));
			hipMalloc(&windowPointer.subject.pointer, BUFFER_SIZE * sizeof(size_t));
                        hipMalloc(&windowPointer.predicate.pointer, BUFFER_SIZE * sizeof(size_t));
                        hipMalloc(&windowPointer.object.pointer, BUFFER_SIZE * sizeof(size_t));
					
			int begin = 0;
			windowPointer.setBegin(begin);
			windowPointer.setEnd(begin);
			windowPointer.setSize(BUFFER_SIZE);
			
			manager.setDevicePointer(windowPointer);
			basic_fnv_1< 1099511628211u, 14695981039346656037u> h_func;
		        //set Queries (select that will be joined)
		        tripleContainer h_queryVector1;
			h_queryVector1.subject = 0;
			h_queryVector1.predicate = 0;
			h_queryVector1.object =  h_func("<http://example.org/int/99>");
			
			
		        tripleContainer h_queryVector2;	
			h_queryVector2.subject = h_func("<http://example.org/int/0>");
			h_queryVector2.predicate = 0;
			h_queryVector2.object = 0;

			cout << "query is: first obj: " << h_queryVector1.object << "; secons subj: " << h_queryVector2.subject << endl;

		        mem_t<tripleContainer> d_queryVector1(1, context);
			hipMemcpy(d_queryVector1.data(), &h_queryVector1, sizeof(tripleContainer), hipMemcpyHostToDevice);
		
		        mem_t<tripleContainer> d_queryVector2(1, context);
			hipMemcpy(d_queryVector2.data(), &h_queryVector2, sizeof(tripleContainer), hipMemcpyHostToDevice);
	
			//set select mask operation
			std::vector<tripleContainer*> selectQuery;
			selectQuery.push_back(d_queryVector1.data());
			selectQuery.push_back(d_queryVector2.data());

			SelectArr arr1 = SelectArr::O;
			SelectArr arr2 = SelectArr::S;
		

			//Creat operation object to pass to query manager
			SelectOperation  selectOp1(&d_queryVector1, arr1, "?s ?p");
			SelectOperation  selectOp2(&d_queryVector2, arr2, "?p ?o");
		
			JoinOperation  joinOp(selectOp1.getResultAddress(), selectOp2.getResultAddress(), "?p");
		
			std::vector<SelectOperation*> selectOperations;
			std::vector<JoinOperation*> joinOperations;
		
			selectOperations.push_back(&selectOp1);
			selectOperations.push_back(&selectOp2);
			joinOperations.push_back(&joinOp);
			
			int stepCount = 100000;
			//std::cout << "starting tmsp " << manager.getTimestampPointer().begin << std::endl;
			
			
			
			/*TimeQuery count(selectOperations, joinOperations, windowPointer, manager.getTimestampPointer(), 5000, 5000);
			manager.addTimeQuery(count);


			TimeQuery count5(selectOperations, joinOperations, windowPointer, manager.getTimestampPointer(), 7000, 7000);
			manager.addTimeQuery(count5);*/
			
			CountQuery count2(selectOperations, joinOperations, windowPointer, 50000);
			manager.addCountQuery(count2);
			
			
			/*CountQuery count3(selectOperations, joinOperations, windowPointer, 50000);
			manager.addCountQuery(count3);
			
			CountQuery count4(selectOperations, joinOperations, windowPointer, 34652);
			manager.addCountQuery(count4);	*/		
			
			gettimeofday(&beginEx, NULL);	

			manager.start();


			
			hipDeviceSynchronize();
			gettimeofday(&end, NULL);


			float exTime = (end.tv_sec - beginEx.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginEx.tv_usec) / 1000 ;
			float prTime = (end.tv_sec - beginPr.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginPr.tv_usec) / 1000 ;
			float cuTime = (end.tv_sec - beginCu.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginCu.tv_usec) / 1000 ;
			

			
			timeCuVector.push_back(cuTime);

						
			cout << "Total time: " << prTime << endl;
			cout << "Cuda time: " << cuTime << endl;
			cout << "Execution time: " << exTime << endl;					
			cout << "" << endl;


			hipFree(windowPointer.subject.pointer);
			hipFree(windowPointer.predicate.pointer);
			hipFree(windowPointer.object.pointer);
			hipFree(windowPointer.rdfStore.pointer);
		}

	
		std::vector<float> statistics;
		
		statistics = stats<float, float>(timeCuVector);	
                cout << "mean cuda time " << statistics[0] << endl;
                cout << "variance cuda time " << statistics[1] << endl;

              /*  statistics = stats<long int, double>(timeExVector);
                cout << "mean ex time " << statistics[0] << endl;
                cout << "variance ex time " << statistics[1] << endl;*/


		cout << "FINAL VALUE IS " << VALUE << std::endl;;
		
		long int sum = 0;
		
		for (int i = 0; i < timeCuVector.size(); i++) {
			std::cout<< "time are " << timeCuVector[i] << std::endl;
		}
		

		
                return 0;
}

