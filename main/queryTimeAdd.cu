#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <unistd.h>

#include <moderngpu/kernel_compact.hxx>
#include <moderngpu/kernel_join.hxx>
#include <moderngpu/kernel_mergesort.hxx>
#include <sys/time.h>

using namespace mgpu;

//struct to contains a single triple with int type.
struct tripleContainer {
        int subject;
        int predicate;
        int object;
};

template<typename type_t>
struct circularBuffer {
	type_t* pointer;
	int begin;
	int end;
	int size;
};

template<typename rdf_t, typename arr_t>
struct triplePointer {
	rdf_t rdfStore;
	arr_t subject;
	arr_t predicate;
	arr_t object;
};

int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}

/*
* Join enum to define order and which element to join
* NJ indicates a non-join value, so it is ignored during join and sorting
* So that it improves performance avoiding uneecessary conditional expression
*/
enum class JoinMask {NJ = -1, SBJ = 0, PRE = 1, OBJ = 2};


//Section for defining operation classes
class JoinOperation 
{	
	private:
		mem_t<tripleContainer>** innerTable;
		mem_t<tripleContainer>** outerTable;
		mem_t<tripleContainer>* innerResult = 0;
		mem_t<tripleContainer>* outerResult = 0;
		
		JoinMask innerMask[3];
		JoinMask outerMask[3];

	public:
		JoinOperation(mem_t<tripleContainer>** innerTable, mem_t<tripleContainer>** outerTable, JoinMask innerMask[3], JoinMask outerMask[3]) {
			this->innerTable = innerTable;
			this->outerTable = outerTable;
			std::copy(innerMask, innerMask + 3, this->innerMask);
			std::copy(outerMask, outerMask + 3, this->outerMask);
		};
			
		mem_t<tripleContainer>** getInnerTable() {
			return this->innerTable;
		};
		
		mem_t<tripleContainer>** getOuterTable() {
			return this->outerTable;
		};
		
		JoinMask* getInnerMask() {
			return this->innerMask;
		};
		
		JoinMask* getOuterMask() {
			return this->outerMask;
		};
		
		mem_t<tripleContainer>* getInnerResult() {
			return this->innerResult;
		};
		
		void setInnerResult(mem_t<tripleContainer>* result) {
			this->innerResult = result;
		};
		
		mem_t<tripleContainer>** getInnerResultAddress() {
			return &innerResult;
		}
		
		mem_t<tripleContainer>* getOuterResult() {
			return this->outerResult;
		};
		
		void setOuterResult(mem_t<tripleContainer>* result) {
			this->outerResult = result;
		};
		
		mem_t<tripleContainer>** getOuterResultAddress() {
			return &outerResult;
		}		
};

enum class SelectArr { S = 0, P = 1, O = 2, SP = 3, SO = 4, PO = 5, SPO = 6};

class SelectOperation 
{

	private:
		mem_t<tripleContainer>* query;
		mem_t<tripleContainer>* result = 0;
		int arr;

	public:
		SelectOperation(mem_t<tripleContainer>* query, SelectArr arr) {
			this->query = query;	
			this->arr = static_cast<int> (arr);
		};

		int getArr() {
			return this-> arr;
		}
			
		mem_t<tripleContainer>* getQuery() {
			return this->query;
		};
		                                                                            
		mem_t<tripleContainer>* getResult() {
			return this->result;
		};
		
		void setResult(mem_t<tripleContainer>* result) {
			this->result = result;
		};
		
		mem_t<tripleContainer>** getResultAddress() {
			return &result;
		}
};


class Query {
	public:
		std::vector<SelectOperation*> select;
		std::vector<JoinOperation*> join;
		triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> windowPointer;

	
		Query(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join, 
			triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> rdfPointer) {

			this->join = join;
			this->select = select;
			this->windowPointer = rdfPointer;
		}

		std::vector<SelectOperation*> getSelect() {
			return select;
		}

		std::vector<JoinOperation*> getJoin() {
			return join;
		}

		void advancePointer(int step) {
			int newEnd = (windowPointer.rdfStore.end + step) % windowPointer.rdfStore.size;
			
			windowPointer.rdfStore.end = newEnd;
			windowPointer.subject.end = newEnd;
			windowPointer.predicate.end = newEnd;
			windowPointer.object.end = newEnd;
		}

		triplePointer<tripleContainer*, int*> getStorePointer() {
			triplePointer<tripleContainer*, int*> pointer;
			pointer.rdfStore = windowPointer.rdfStore.pointer;
			pointer.subject = windowPointer.subject.pointer;
			pointer.predicate = windowPointer.predicate.pointer;
			pointer.object = windowPointer.object.pointer;
			
			return pointer;
		}
	
};

void queryManager(std::vector<SelectOperation*> selectOp, std::vector<JoinOperation*> joinOp, 
		triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> d_pointer, 
		const int storeSize);

class CountQuery : public Query {
	private:
		int count;
		int currentCount;

	public:
		CountQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> rdfPointer,
			int count) : Query(select, join, rdfPointer) {

				this->count = count;
				this->currentCount = 0;
		} 

		void incrementCount() {
			this->currentCount++;
		}

		bool isReady() {
			if (currentCount == count) {
				count = 0;
				return true;
			}
			return false;
		}
	
		void launch() {
			queryManager(getSelect(), getJoin(), windowPointer, count);
			int newBegin = (windowPointer.rdfStore.begin + count) % windowPointer.rdfStore.size;
			windowPointer.rdfStore.begin = newBegin;
			windowPointer.subject.begin = newBegin;
			windowPointer.predicate.begin = newBegin;
			windowPointer.object.begin = newBegin;
 
		}
};

class TimeQuery : public Query {
	private:
		circularBuffer<long int> timestampPointer;
		int stepTime;
		int windowTime;
		long int lastTimestamp;
		bool isFirst;

	public:
		TimeQuery(std::vector<SelectOperation*> select, std::vector<JoinOperation*> join,
			triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> rdfPointer,
			circularBuffer<long int> timestampPointer,
			int windowTime, int stepTime) : Query(select, join, rdfPointer) {
				this->stepTime = stepTime;
				this->windowTime = windowTime;
				this->lastTimestamp = 0;
				this->isFirst = 0;
				this->timestampPointer = timestampPointer;
		}
		
		void setStartingTimestamp(long int timestamp) {
			this->lastTimestamp = timestamp;
		}

		void advancePointer(int step) : advancePointer(int) {
			timestampPointer.end = (windowPointer.timestampPointer.end + size) % windowPointer.timestampPointer.size;
		}

		

		bool isReady(long int newTimestamp) {
			if (isFirst) {
				if (lastTimestmp + windowTime <= newTimestamp) {
					lastTimestamp += spanTime;
					return true;
				}

			} else {
				if (lastTimestamp + stepTime <= newTimestamp {
					lastTimestamp += spanTime;
					return true;
			}
			
			return false;
		}

		long int* getTimestampPointer() {
			return this->timestampPointer.pointer;
		}

		void launch() {
			
			if (!isFirst) {
				long int startingTime = lastTimestamp - (long int) windowTime;					
				int newBegin = 0;

				if (timestampPointer.end < timestampPointer.begin) {
					bool found = false;

					for(int i = timestampPointer.begin; i < timestampPointer.size; i++) {
						if (timestampPointer.pointer[i] >= startingTime) {
							found = true;
							newBegin = i;
							break;
						}
					}				

					if (found == false) {
						for(int i = 0; i < timestampPointer.end; i++) {
							if (timestampPointer.pointer[i] >= startingTime) {
								newBegin = i;
								break;
							}				
						}				
					}

				} else {
					for (int i = 0; i < timestampPointer.end; i++) {
						if (timestampPointer.pointer[i] >= startingTime) {
							newBegin = i;
							break;
						}
					}

				}

				windowPointer.rdfStore.begin = newBegin;
				windowPointer.subject.begin = newBegin;
				windowPointer.predicate.begin = newBegin;
				windowPointer.object.begin = newBegin;

			} else {
				isFirst = false;
			}	

			

			int diff = windowPointer.rdfStore.end - windowPointer.rdfStore.begin;
			int currentSize = (diff >= 0 ? diff : windowPointer.rdfStore.size + diff); 
			queryManager(getSelect(), getJoin(), windowPointer, currentSize);  
		}


};


class TripleGenerator {
	private:
		int spanTime;
		std::ifstream rdfStoreFile;

		std::vector<tripleContainer> rdfBuffer;
		std::vector<int> subjectBuffer;
		std::vector<int> predicateBuffer;
		std::vector<int> objectBuffer;

		std::vector<CountQuery> countQueries;
		std::vector<TimeQuery> timeQueries;

		circularBuffer<long int> timestampPointer;
		triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> devicePointer;

	public:
		void checkStep();
		void advanceDevicePointer();
		
		TripleGenerator(int spanTime, std::ifstream inputFile)   {
			this->spanTime = spanTime;
			this->rdfStoreFile = inputFile; 
			long int* timestamp = (long int*) malloc(buffSize * sizeof(long int));
		}
		
		void start() {
			std::string strInput;
 
			while (std::getline(rdfStoreFile, strInput)) {
				std::vector<std::string> triple;
		                separateWords(strInput, triple, ' ');

				tripleContainer currentTriple  = {atoi(triple[0].c_str()), atoi(triple[1].c_str()), atoi(triple[2].c_str())};
				
				struct timeval tp;
				gettimeofday(&tp, NULL);
				long int ms = tp.tv_sec * 1000 + tp.tv_usec / 1000;

				timestapPointer.pointer[timestampPointer.end] = ms;
				timestampPointer.end = (windowPointer.timestampPointer.end + 1) % windowPointer.timestampPointer.size;
				timestampPointer.begin = timestampPointer.end;

				rdfBuffer.push_back(currentTriple);
				subjectBuffer.push_back(currentTriple.subject);
				predicateBuffer.push_back(currentTriple.predicate);
				objectBuffer.push_back(currentTriple.object);

				checkStep();
				
				usleep(spanTime);
			}			
		}

		void advanceDevicePointer() {
			int copySize = rdfBuffer.size();
			
			circularBuffer<tripleContainer> rdfBuff = devicePointer.rdfStore;

			int newEnd = (rdfBuff.end + copySize) % rdfBuff.size;

			if (newEnd < rdfBuff.end) {
				int finalHalf = rdfBuff.size - rdfBuff.end;
				hipMemcpy(devicePointer.rdfStore.end + devicePointer.rdfStore.pointer, rdfBuffer, finalHalf, hipMemcpyHostToDevice); 
				hipMemcpy(devicePointer.subject.end + devicePointer.subject.pointer, subjectBuffer, finalHalf, hipMemcpyHostToDevice);
				hipMemcpy(devicePointer.predicate.end + devicePointer.predicate.pointer, predicateBuffer, finalHalf, hipMemcpyHostToDevice);
				hipMemcpy(devicePointer.object,end + devicePointer.object.pointer, objectBuffer, finalHalf, hipMemcpyHostToDevice);			
	
				int firstHalf = copySize - finalHalf;
				hipMemcpy(devicePointer.rdfStore.pointer, rdfBuffer, firstHalf, hipMemcpyHostToDevice); 
				hipMemcpy(devicePointer.subject.pointer, subjectBuffer, firstHalf, hipMemcpyHostToDevice);
				hipMemcpy(devicePointer.predicate.pointer, predicateBuffer, firstHalf, hipMemcpyHostToDevice);
				hipMemcpy(devicePointer.object.pointer, objectBuffer, firstHalf, hipMemcpyHostToDevice);			
			} else {
				hipMemcpy(devicePointer.rdfStore.end + devicePointer.rdfStore.pointer, rdfBuffer, copySize, hipMemcpyHostToDevice); 
				hipMemcpy(devicePointer.subject.end + devicePointer.subject.pointer, subjectBuffer, copySize, hipMemcpyHostToDevice);
				hipMemcpy(devicePointer.predicate.end + devicePointer.predicate.pointer, predicateBuffer, copySize, hipMemcpyHostToDevice);
				hipMemcpy(devicePointer.object,end + devicePointer.object.pointer, objectBuffer, copySize, hipMemcpyHostToDevice);			
			}


			devicePointer.rdfStore.end = newEnd;
			devicePointer.subject.end = newEnd;
			devicePointer.predicate.end = newEnd;
			devicePointer.object.end = newEnd;

			rdfBuffer.clear();
			subjectBuffer.clear();
			predicateBuffer.clear();
			objectBuffer.clear();
		}
		
		void checkStep() {
			
			for (CountQuery query : countQueries)  {
				query.incrementCount();
				if (query.isReady()) {
					advanceDevicePointer();
					int copySize = rdfBuffer.size();
			
					for (CountQuery other : countQueries)  {
						other.advancePointer(copySize);
					}
				
					for (TimeQuery other : timeQueries) {
						other.advancePointer(copySize);
					}

					query.launch();
				}

			}

			for (TimeQuery query : timeQueries) {
				if (query.isRedy(timestamps.pointer[timestamps.end - 1])) {
					advanceDevicePointer();
					int copySize = rdfBuffer.size();
			
					for (CountQuery query : countQueries)  {
						query.advancePointer(copySize);
					}
				
					for (TimeQuery query : timeQueries) {
						query.advancePointer(copySize);
					}

					query.launch();

				}
			}
		}



};


__global__ void unarySelect (circularBuffer<int> src, int* value, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size, int storeSize) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (abs(src.end - src.begin +  src.size) % src.size) ) {
		return;
	}	

	int newIndex = (src.begin + index) % src.size;
	
	if (src.pointer[newIndex] == (*value)) {
		int add = atomicAdd(size, 1);
		dest[add] = store.pointer[newIndex];
	}
}

__global__ void binarySelect (circularBuffer<int> src1, circularBuffer<int> src2, int* value1, int* value2, tripleContainer* dest, circularBuffer<tripleContainer> store, int* size, int storeSize) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= (abs(src1.end - src1.begin + src1.size) % src1.size) ) {
		return;
	}		

	int newIndex = (src1.begin + index) % src1.size;
	if ((src1.pointer[newIndex] == (*value1)) && (src2.pointer[newIndex] == (*value2))) {
		int add = atomicAdd(size, 1);
		dest[add] = store.pointer[newIndex];
	}
}



/*
* Make multiple select query, with specified comparison condition,
* on a triple store. Both queries and the store are supposed to 
* be already on the device. 
* 
* @param d_selectQueries : the array in which are saved the select values
* @param d_storePointer : pointer on the device to the triple store
* @param storeSize : size of the triple store
* @param comparatorMask : array of triple of comparator that are applied to the queries
*			must be of the size of the d_selectQueries
* @return a vector of type mem_t in which are saved the query results.
*/
std::vector<mem_t<tripleContainer>*> rdfSelect(const std::vector<tripleContainer*> d_selectQueries, 
		triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> d_pointer,
		const int storeSize, 
		std::vector<int*> comparatorMask,
		std::vector<int>  arrs) 
{
	standard_context_t context;
	//Initialize elements
	int querySize =  d_selectQueries.size();
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	int* currentSize;
	hipMalloc(&currentSize, sizeof(int));
	int* zero = (int*) malloc(sizeof(int));
	*zero = 0;

	int* finalResultSize  = (int*) malloc(sizeof(int));

	//Cycling on all the queries
	for (int i = 0; i < querySize; i++) {
		//Save variable to pass to the lambda operator
		tripleContainer* currentPointer = d_selectQueries[i];
		
		mem_t<tripleContainer>* currentResult = new mem_t<tripleContainer>(storeSize, context);

		int gridSize = 300;
	        int blockSize = (storeSize / gridSize) + 1;
		hipMemcpy(currentSize, zero, sizeof(int), hipMemcpyHostToDevice);
			
		switch(arrs[i]) {

			case(0): {
				int* value = &(currentPointer->subject);

				unarySelect<<<gridSize,blockSize>>>(d_pointer.subject, value, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}

			case(1): {
				int* value = &(currentPointer->predicate);
			
				unarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, value, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}
						
			case(2): {
                                int* value = &(currentPointer->object);

                                unarySelect<<<gridSize,blockSize>>>(d_pointer.object, value, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);
                                break;
			}
			
			case(3): {
				int* value1 = &(currentPointer->subject);
				int* value2 = &(currentPointer->predicate);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.predicate, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}

			case(4): {
				int* value1 = &(currentPointer->subject);
				int* value2 = &(currentPointer->object);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.subject, d_pointer.object, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}

			case(5): {
				int* value1 = &(currentPointer->predicate);
				int* value2 = &(currentPointer->object);

				binarySelect<<<gridSize,blockSize>>>(d_pointer.predicate, d_pointer.object, value1, value2, currentResult->data(), d_pointer.rdfStore, currentSize, storeSize);

				break;
			}
						
			case(6): {
				hipMemcpy(currentResult->data(), d_pointer.rdfStore.pointer, storeSize * sizeof(tripleContainer), hipMemcpyDeviceToDevice);
				hipMemcpy(currentSize, &storeSize, sizeof(int), hipMemcpyHostToDevice);
                                break;
			}
			
			
			default: {
				printf("ERROR ERRROR ERROR ERROR ERROR ERROR ERROR");
			}


		}
		
                hipMemcpy(finalResultSize, currentSize, sizeof(int), hipMemcpyDeviceToHost);
		currentResult->setSize(*finalResultSize);
		finalResults.push_back(currentResult);
	}
	hipFree(currentSize);
	
	return finalResults;
}



//Sorter for sorting the triple due to theorder defined by the sortMask
 class TripleSorter {
	private:
		int sortMask[3];
	public:
		TripleSorter(JoinMask sortMask[3]) {
			this->sortMask[0] = static_cast<int> (sortMask[0]);
			this->sortMask[1] = static_cast<int> (sortMask[1]);
			this->sortMask[2] = static_cast<int> (sortMask[2]);
				
		}
		
		MGPU_DEVICE bool operator() (tripleContainer a, tripleContainer b) {
			int tripleA[3] = {a.subject, a.predicate, a.object};
			int tripleB[3] = {b.subject, b.predicate, b.object};
			
			if ((sortMask[0] != -1) && (tripleA[sortMask[0]] < tripleB[sortMask[0]])) {
				return true;
			}
			
			if ((sortMask[1] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] < tripleB[sortMask[1]])) {
				return true;
			}
			
			if ((sortMask[2] != -1) && (tripleA[sortMask[0]] == tripleB[sortMask[0]]) && (tripleA[sortMask[1]] == tripleB[sortMask[1]]) && (tripleA[sortMask[2]] < tripleB[sortMask[2]])) {
				return true;
			}
			
			return false;
		}
};


//Computes inner < outer when doing join to find the elements that needs to be joined
class TripleComparator
{
	private:
		int maskA[3];
		int maskB[3];
	public:
		TripleComparator(JoinMask innerMask[3], JoinMask outerMask[3]) {
			maskA[0] = static_cast<int> (innerMask[0]);
			maskA[1] = static_cast<int> (innerMask[1]);
			maskA[2] = static_cast<int> (innerMask[2]);
			
			maskB[0] = static_cast<int> (outerMask[0]);
			maskB[1] = static_cast<int> (outerMask[1]);
			maskB[2] = static_cast<int> (outerMask[2]);			
		}
		
		MGPU_DEVICE bool operator() (tripleContainer a, tripleContainer b) {			
			int tripleA[3] = {a.subject, a.predicate, a.object};
			int tripleB[3] = {b.subject, b.predicate, b.object};

			if ((maskA[0] != -1) && (tripleA[maskA[0]] < tripleB[maskB[0]])) {
				return true;
			}
		
			if ((maskA[1] != -1) && (tripleA[maskA[0]] == tripleB[maskB[0]]) && (tripleA[maskA[1]] < tripleB[maskA[1]])) {
				return true;
			}
			
			if ((maskA[2] != -1) && (tripleA[maskA[0]] == tripleB[maskB[0]]) && (tripleA[maskA[1]] == tripleB[maskA[1]]) && (tripleA[maskA[2]] < tripleB[maskA[2]])) {
				return true;
			}
		
			return false;
		}
};


__global__ void indexCopy(tripleContainer* innerSrc, tripleContainer* innerDest, tripleContainer* outerSrc, tripleContainer* outerDest, int2* srcIndex, int maxSize) 
{
	int destIndex = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (destIndex  >= maxSize)  {
		return;
	}
	
	//INVETERD INDEX DUE TO INVERTED JOIN PROBLEM (it should be inner = x, outer = y)
	int innerIndex = srcIndex[destIndex].y;
	int outerIndex = srcIndex[destIndex].x;
	
	innerDest[destIndex] = innerSrc[innerIndex];	
	outerDest[destIndex] = outerSrc[outerIndex];
}

std::vector<mem_t<tripleContainer>*> rdfJoin(tripleContainer* innerTable, int innerSize, tripleContainer* outerTable, int outerSize, JoinMask innerMask[3], JoinMask outerMask[3])
{
	standard_context_t context;
	std::vector<mem_t<tripleContainer>*> finalResults;
	
	TripleSorter* innerSorter = new TripleSorter(innerMask);
	TripleSorter* outerSorter = new TripleSorter(outerMask);
	
	//Sort the two input array
	mergesort(innerTable, innerSize , *innerSorter, context);
	mergesort(outerTable, outerSize , *outerSorter, context);
	
	TripleComparator* comparator = new TripleComparator(innerMask, outerMask);
	
	//BUG che mi costringe ad invertire inner con outer?
	mem_t<int2> joinResult = inner_join(outerTable, outerSize, innerTable, innerSize,  *comparator, context);
		
	mem_t<tripleContainer>* innerResults = new mem_t<tripleContainer>(joinResult.size(), context);
        mem_t<tripleContainer>* outerResults = new mem_t<tripleContainer>(joinResult.size(), context);
	
	//SETTARE DIVISIONE CORRETTA
	//BIsogna settare come comporatrsi quando il valore della join supera i 129k risultati
	int gridSize = 64;
	int blockSize = (joinResult.size() / gridSize) + 1; 
	indexCopy<<<gridSize, blockSize>>>(innerTable, innerResults->data(), outerTable, outerResults->data(), joinResult.data(), joinResult.size());

	finalResults.push_back(innerResults);
	finalResults.push_back(outerResults);

	return finalResults;
}



/**
* Function for managing query execution
**/
void queryManager(std::vector<SelectOperation*> selectOp, std::vector<JoinOperation*> joinOp, 
		triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> d_pointer, 
		const int storeSize) {

	std::vector<tripleContainer*> d_selectQueries;
	std::vector<int*> comparatorMask;
	std::vector<int> arrs;

	for (int i = 0; i < selectOp.size(); i++) {
		d_selectQueries.push_back(selectOp[i]->getQuery()->data());
		arrs.push_back(selectOp[i]->getArr());
	}
	

	std::vector<mem_t<tripleContainer>*> selectResults = rdfSelect(d_selectQueries, d_pointer, storeSize, comparatorMask, arrs);

	for (int i = 0; i < selectResults.size(); i++) {
		selectOp[i]->setResult(selectResults[i]);
	}
	
	
	for (int i = 0; i < joinOp.size(); i++) {
		mem_t<tripleContainer>* innerTable = *joinOp[i]->getInnerTable();
		mem_t<tripleContainer>* outerTable = *joinOp[i]->getOuterTable();
		std::vector<mem_t<tripleContainer>*>  joinResult = rdfJoin(innerTable->data(), innerTable->size(), outerTable->data(), outerTable->size(), joinOp[i]->getInnerMask(), joinOp[i]->getOuterMask());
		joinOp[i]->setInnerResult(joinResult[0]);
		joinOp[i]->setOuterResult(joinResult[1]);				
	}
	
}




template<typename type_t, typename accuracy>
std::vector<accuracy> stats(std::vector<type_t> input) {
	int size = input.size();
	float mean = 0;
	float variance = 0;
	for (int i = 0; i < size; i++) {
		mean += (accuracy) input[i];
                variance += (accuracy)  (input[i] * input[i]);
        }
        mean = mean / ((accuracy) size);
        variance = variance / ((accuracy) size);
        variance = variance - (mean * mean);
        std::vector<accuracy> statistic;
	statistic.push_back(mean);
	statistic.push_back(variance);
	return statistic;
}

int main(int argc, char** argv) {
 
		using namespace std;
		struct timeval beginPr, beginCu, beginEx, end;
		gettimeofday(&beginPr, NULL);	
		hipDeviceReset();
		standard_context_t context;

                size_t BUFFER_SIZE = 400000;
		triplePointer<circularBuffer<tripleContainer>, circularBuffer<int>> windowPointer;

		std::vector<float> timeCuVector;                
		std::vector<float> timeExVector;
		std::vector<int> firstVector;
		std::vector<int> secondVector;
		std::vector<int> resultVector;
                int N_CYCLE = 1;
		for (int i = 0; i < N_CYCLE; i++) {
			gettimeofday(&beginCu, NULL);

			hipMalloc(windowPointer.rdfStore.pointer, BUFFER_SIZE * sizeof(tripleContainer));
			hipMalloc(windowPointer.subject.pointer, BUFFER_SIZE * sizeof(int));
                        hipMalloc(windowPointer.predicate.pointer, BUFFER_SIZE * sizeof(int));
                        hipMalloc(windowPointer.object.pointer, BUFFER_SIZE * sizeof(int));
		
		        //set Queries (select that will be joined)
		        tripleContainer h_queryVector1 =  {-1, -1 , 99};
		        tripleContainer h_queryVector2 =	{0, -1, -1}; 

			cout << "query is " << h_queryVector1.object << " " << h_queryVector2.subject << endl;

		        mem_t<tripleContainer> d_queryVector1(1, context);
			hipMemcpy(d_queryVector1.data(), &h_queryVector1, sizeof(tripleContainer), hipMemcpyHostToDevice);
		
		        mem_t<tripleContainer> d_queryVector2(1, context);
			hipMemcpy(d_queryVector2.data(), &h_queryVector2, sizeof(tripleContainer), hipMemcpyHostToDevice);
	
			//set select mask operation
			std::vector<tripleContainer*> selectQuery;
			selectQuery.push_back(d_queryVector1.data());
			selectQuery.push_back(d_queryVector2.data());

			SelectArr arr1 = SelectArr::O;

			SelectArr arr2 = SelectArr::S;
			
		
			//set Join mask
			JoinMask innerMask[3];
			innerMask[0] = JoinMask::PRE;
			innerMask[1] = JoinMask::NJ;
			innerMask[2] = JoinMask::NJ;
			
			JoinMask outerMask[3];
			outerMask[0] = JoinMask::PRE;
			outerMask[1] = JoinMask::NJ;
			outerMask[2] = JoinMask::NJ;

			//Creat operation object to pass to query manager
			SelectOperation  selectOp1(&d_queryVector1, arr1);
			SelectOperation  selectOp2(&d_queryVector2, arr2);
		
			JoinOperation  joinOp(selectOp1.getResultAddress(), selectOp2.getResultAddress(), innerMask, outerMask);
		
			std::vector<SelectOperation*> selectOperations;
			std::vector<JoinOperation*> joinOperations;
		
			selectOperations.push_back(&selectOp1);
			selectOperations.push_back(&selectOp2);
			joinOperations.push_back(&joinOp);
		
			gettimeofday(&beginEx, NULL);	
			
			queryManager(selectOperations, joinOperations, buffer->getPointer(), fileLength);
			
			//Retrive results from memory
			std::vector<tripleContainer> finalInnerResults = from_mem(*joinOp.getInnerResult());
			std::vector<tripleContainer> finalOuterResults = from_mem(*joinOp.getOuterResult());
			
			hipDeviceSynchronize();
			gettimeofday(&end, NULL);

                        std::vector<tripleContainer> selectResults = from_mem(*selectOp1.getResult());
                        std::vector<tripleContainer> selectResults2 = from_mem(*selectOp2.getResult());

			float exTime = (end.tv_sec - beginEx.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginEx.tv_usec) / 1000 ;
			float prTime = (end.tv_sec - beginPr.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginPr.tv_usec) / 1000 ;
			float cuTime = (end.tv_sec - beginCu.tv_sec ) * 1000 + ((float) end.tv_usec - (float) beginCu.tv_usec) / 1000 ;
			
			timeCuVector.push_back(cuTime);
			timeExVector.push_back(exTime);
			firstVector.push_back(selectResults.size());
			secondVector.push_back(selectResults2.size());
			resultVector.push_back(finalOuterResults.size());

			/*
			//Print Results
			cout << "first select result" << endl;
			for (int i = 0; i < selectResults.size(); i++) {
				cout << selectResults[i].subject << " " << selectResults[i].predicate << " "  << selectResults[i].object << endl; 
			}
		
			cout << "second select result" << endl;
			for (int i = 0; i < selectResults2.size(); i++) {
				cout << selectResults2[i].subject << " " << selectResults2[i].predicate << " "  << selectResults2[i].object << endl; 
			}
		
			cout << "final inner result" << endl;
			for (int i = 0; i < finalInnerResults.size(); i++) {
			cout << finalInnerResults[i].subject << " " << finalInnerResults[i].predicate << " "  << finalInnerResults[i].object << endl; 
			} 
			
			cout << "final inner result" << endl;
			for (int i = 0; i < finalOuterResults.size(); i++) {
				cout << finalOuterResults[i].subject << " " << finalOuterResults[i].predicate << " "  << finalOuterResults[i].object << endl; 
			} */
			
			//Print current cycle results
			cout <<"first Select Size " << selectResults.size() << endl;
			cout << "second Select Size " << selectResults2.size() << endl;
			cout << "join Size " << finalOuterResults.size() << endl;
						
			cout << "Total time: " << prTime << endl;
			cout << "Cuda time: " << cuTime << endl;
			cout << "Execution time: " << exTime << endl;					
			cout << "" << endl;

			hipFree((*joinOp.getInnerResult()).data());
			hipFree((*joinOp.getOuterResult()).data());
			hipFree((*selectOp1.getResult()).data());
			hipFree((*selectOp2.getResult()).data());
			hipFree(d_storeVector);
			hipFree(d_storeVector);
			hipFree(d_subject);
			hipFree(d_object);
			hipFree(d_predicate);
		}
		
		std::vector<float> statistics;
		
		statistics = stats<float, float>(timeCuVector);	
                cout << "mean cuda time " << statistics[0] << endl;
                cout << "variance cuda time " << statistics[1] << endl;

                statistics = stats<float, float>(timeExVector);
                cout << "mean ex time " << statistics[0] << endl;
                cout << "variance ex time " << statistics[1] << endl;
/*
                statistics = stats<int, longlong>(firstVector);
                cout << "mean first select size " << statistics[0] << endl;
                cout << "variance first select size " << statistics[1] << endl;

                statistics = stats<int, longlong>(secondVector);
                cout << "mean second select size " << statistics[0] << endl;
                cout << "variance second select size " << statistics[1] << endl;
	 
                statistics = stats<int, longlong>(resultVector);
                cout << "mean join size " << statistics[0] << endl;
                cout << "variance join size " << statistics[1] << endl;
*/
                return 0;

}



