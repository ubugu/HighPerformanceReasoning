
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <moderngpu/kernel_compact.hxx>

//Define callable function from host and deivce 
#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif

using namespace mgpu;

//struct to contains a single triple with a element_t type.
template<typename element_t>
struct tripleContainer {
        element_t subject;
        element_t predicate;
        element_t object;
};



int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}

int main(int argc, char** argv) {
/*
                const int FILE_LENGHT = 100000;
                size_t rdfSize = FILE_LENGHT * sizeof(tripleContainer);

                tripleContainer* h_rdfStore = (tripleContainer*) malloc(rdfSize);

                //read store from rdfStore
                ifstream rdfStoreFile ("../rdfStore/rdfSorted.txt");

                string strInput;

                for (int i = 0; i < FILE_LENGHT; i++) {
                        getline(rdfStoreFile,strInput);

                        std::vector<string> triple ;
                        separateWords(strInput, triple, ' ');

                        h_rdfStore[i] =  {atoi(triple[0].c_str()), atoi(triple[1].c_str()), atoi( triple[2].c_str())};
                }
                rdfStoreFile.close();

                        h_rdfStore[i] =  {atoi(triple[0].c_str()), atoi(triple[1].c_str()), atoi( triple[2].c_str())};
                }
                rdfStoreFile.close();


                //take query parameters
                const int TUPLE_LENGHT = 4;
                int queryLenght = (argc - 1) / TUPLE_LENGHT;

                size_t querySize = queryLenght * sizeof(tripleContainer);
                tripleContainer* h_queryVector = (tripleContainer*) malloc(querySize);

                for (int i = 0; i < queryLenght; i++) {
                        int index = 1 + i *  TUPLE_LENGHT;
                        h_queryVector[i] = {atoi(argv[index]), atoi(argv[index + 1]), atoi(argv[index + 2])};
                        cout << atoi(argv[index]) << endl;
                }
*/
	return 0;
}


//Make a select query on the store pointer saved on the
template<typename element_t>
int select(const std::vector<tripleContainer<elemnt_t>*> d_selectQueries, const tripleContainer<elemnt_t>* d_storePointer, const int storeSize, const element_t varaibleToken) {

                return 0;
		int querySize =  d_selectQueries.size();
		
		standard_context_t context;		
/*
		
		for (int i = 0; i < querySize; i++) {
			tripleContainer<element_t> currentPointer = d_selectQueries[i]; 
	             
			int query_count = compact.upsweep([=] MGPU_DEVICE(int index) {
				bool subjectEqual = false;
				bool predicateEqual = false;
				bool objectEqual = false;

				if (currentPointer->subject != variableToken) {
					subjectEqual = (currentPointer->subject == ter[index].subject);
				} else {
					subjectEqual = true;
				}

				if (currentPointer->predicate != variableToken) {
				predicateEqual = (currentPointer->predicate == rdfStorePointer[index].predicate);
				} else {
				predicateEqual = true;
				}
	
				if (currentPointer->object != varaibleToken) {
					objectEqual = (currentPointer->object == rdfStorePointer[index].object);
				} else {
					objectEqual = true;
				}

				return (subjectEqual && predicateEqual && objectEqual);
			});

			tripleContainer<element_t>* d_result;
			hipMalloc(&d_result, query_count);

			compact.downsweep([=] MGPU_DEVICE(int dest_index, int source_index) {
				d_result[dest_index] = rdfStorePointer[source_index];
			});


		}
		
*/
		
	
	
}
