#include <iostream>
#include <fstream>
#include <cstdlib>
#include <moderngpu/kernel_compact.hxx>

//Define callable function from host and deivce 
#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif

using namespace mgpu;

//struct to contains a single triple with a element_t type.
template<typename element_t>
struct tripleContainer {
        element_t subject;
        element_t predicate;
        element_t object;
};

enum comparator_t {
	LT,
	LEQ,
	EQ,
	GT,
	GEQ,
	NN
};

template<typename comparator>
std::function<bool()>  lambdaSelectCreator(bool negate)
{
	comparator compare[3];

	if (negate == true) {
		compare = !comparator();
	} else {
		compare = comparator();
	}

	return [=](){return compare(1,2);};
}

int separateWords(std::string inputString, std::vector<std::string> &wordVector,const char separator ) {	
	const size_t zeroIndex = 0;
	size_t splitIndex = inputString.find(separator);
	
	while (splitIndex != -1)
		{
			wordVector.push_back(inputString.substr(zeroIndex, splitIndex));	
			inputString = inputString.substr(splitIndex + 1 , inputString.length() - 1);
			splitIndex = inputString.find(separator);
		}
	
	wordVector.push_back(inputString);
	return 0;
}




//Make a select query on the store pointer saved on the
template<typename element_t>
std::vector<int> rdfSelect(const std::vector<tripleContainer<element_t>*> d_selectQueries, const tripleContainer<element_t>* d_storePointer, const int storeSize, const element_t variableToken, std::vector<tripleContainer<element_t>*>** result) {

		int querySize =  d_selectQueries.size();
		
		standard_context_t context;		
		auto compact = transform_compact(storeSize, context);
		std::vector<tripleContainer<element_t>*> finalResultPointers;
		std::vector<int> resultsSize;
	
		for (int i = 0; i < querySize; i++) {
			tripleContainer<element_t>* currentPointer = d_selectQueries[i]; 
	             
			int query_count = compact.upsweep([=] MGPU_DEVICE(int index) {
				bool subjectEqual = false;
				bool predicateEqual = false;
				bool objectEqual = false;

				if (currentPointer->subject != variableToken) {
					subjectEqual = (currentPointer->subject == d_storePointer[index].subject);
				} else {
					subjectEqual = true;
				}

				if (currentPointer->predicate != variableToken) {
				predicateEqual = (currentPointer->predicate == d_storePointer[index].predicate);
				} else {
				predicateEqual = true;
				}
	
				if (currentPointer->object != variableToken) {
					objectEqual = (currentPointer->object == d_storePointer[index].object);
				} else {
					objectEqual = true;
				}

				return (subjectEqual && predicateEqual && objectEqual);
			});

			tripleContainer<element_t>* d_result;
			hipMalloc(&d_result, query_count);

			compact.downsweep([=] MGPU_DEVICE(int dest_index, int source_index) {
				d_result[dest_index] = d_storePointer[source_index];
			});
			
			finalResultPointers.push_back(d_result);
			resultsSize.push_back(query_count);
		}

		*result = &finalResultPointers;
		return resultsSize;
}	


int main(int argc, char** argv) {
		using namespace std;
		if (argc < 4 ) {
			std::cout << "errore " << endl;
		}


		return 0;

                const int FILE_LENGHT = 100000;
                size_t rdfSize = FILE_LENGHT * sizeof(tripleContainer<int>);
                tripleContainer<int>* h_rdfStore = (tripleContainer<int>*) malloc(rdfSize);

                //read store from rdfStore
                ifstream rdfStoreFile ("../rdfStore/rdfSorted.txt");

                string strInput;

                for (int i = 0; i < FILE_LENGHT; i++) {
                        getline(rdfStoreFile,strInput);

                        std::vector<string> triple ;
                        separateWords(strInput, triple, ' ');

                        h_rdfStore[i] =  {atoi(triple[0].c_str()), atoi(triple[1].c_str()), atoi( triple[2].c_str())};
                }
                rdfStoreFile.close();

                //take query parameters
                const int TUPLE_LENGHT = 4;
                int queryLenght = (argc - 1) / TUPLE_LENGHT;

                size_t querySize = queryLenght * sizeof(tripleContainer<int>);
                tripleContainer<int>* h_queryVector = (tripleContainer<int>*) malloc(querySize);

                for (int i = 0; i < queryLenght; i++) {
                        int index = 1 + i *  TUPLE_LENGHT;
                        h_queryVector[i] = {atoi(argv[index]), atoi(argv[index + 1]), atoi(argv[index + 2])};
                }

		tripleContainer<int>* d_storeVector;
		hipMalloc(&d_storeVector, rdfSize);
		hipMemcpy(d_storeVector, h_rdfStore, rdfSize, hipMemcpyHostToDevice);		
		tripleContainer<int>* d_queryVector;
		hipMalloc(&d_queryVector, sizeof(tripleContainer<int>));
		hipMemcpy(d_queryVector, h_queryVector, sizeof(tripleContainer<int>), hipMemcpyHostToDevice);
		
		std::vector<tripleContainer<int>*> selectQuery;
		std::vector<int> selectResult;
		selectQuery.push_back(d_queryVector);
		int token = -1;
                std::vector<tripleContainer<int>*>** resultPointer = 0;
		selectResult = rdfSelect<int>(selectQuery, d_storeVector, FILE_LENGHT, token, resultPointer); 
		
	//	hipMemcpy(resultsss, selectResult[0] 
		
		
		return 0;
}


